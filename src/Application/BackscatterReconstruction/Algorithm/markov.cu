#include "hip/hip_runtime.h"

#ifdef _WIN32
#define WIN32_LEAN_AND_MEAN
#endif

#include "markov.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>

#include <Application/BackscatterReconstruction/Algorithm/cuda_common/helper_functions.h>
#include <Application/BackscatterReconstruction/Algorithm/cuda_common/hip/hip_runtime_api.h>
#include <Application/BackscatterReconstruction/Algorithm/cuda_common/cutil_math.h>

// material volumes in the collection
float4 *cudaVolumeLinearCurrent = NULL;
float4 *cudaVolumeLinearCollection = NULL;
hipArray *cudaVolumeArray00 = NULL;
hipArray *cudaVolumeArray01 = NULL;
hipArray *cudaVolumeArray02 = NULL;
hipArray *cudaVolumeArray10 = NULL;
hipArray *cudaVolumeArray11 = NULL;
hipArray *cudaVolumeArray12 = NULL;
hipArray *cudaVolumeArray20 = NULL;
hipArray *cudaVolumeArray21 = NULL;
hipArray *cudaVolumeArray22 = NULL;
texture<float4, hipTextureType3D, hipReadModeElementType> cudaVolumeTextures00;
texture<float4, hipTextureType3D, hipReadModeElementType> cudaVolumeTextures01;
texture<float4, hipTextureType3D, hipReadModeElementType> cudaVolumeTextures02;
texture<float4, hipTextureType3D, hipReadModeElementType> cudaVolumeTextures10;
texture<float4, hipTextureType3D, hipReadModeElementType> cudaVolumeTextures11;
texture<float4, hipTextureType3D, hipReadModeElementType> cudaVolumeTextures12;
texture<float4, hipTextureType3D, hipReadModeElementType> cudaVolumeTextures20;
texture<float4, hipTextureType3D, hipReadModeElementType> cudaVolumeTextures21;
texture<float4, hipTextureType3D, hipReadModeElementType> cudaVolumeTextures22;


// source attenuation volumes in the collection
hipArray *cudaSourceArray00 = NULL;
hipArray *cudaSourceArray01 = NULL;
hipArray *cudaSourceArray02 = NULL;
hipArray *cudaSourceArray10 = NULL;
hipArray *cudaSourceArray11 = NULL;
hipArray *cudaSourceArray12 = NULL;
hipArray *cudaSourceArray20 = NULL;
hipArray *cudaSourceArray21 = NULL;
hipArray *cudaSourceArray22 = NULL;
texture<float, hipTextureType3D, hipReadModeElementType> cudaSourceTextures00;
texture<float, hipTextureType3D, hipReadModeElementType> cudaSourceTextures01;
texture<float, hipTextureType3D, hipReadModeElementType> cudaSourceTextures02;
texture<float, hipTextureType3D, hipReadModeElementType> cudaSourceTextures10;
texture<float, hipTextureType3D, hipReadModeElementType> cudaSourceTextures11;
texture<float, hipTextureType3D, hipReadModeElementType> cudaSourceTextures12;
texture<float, hipTextureType3D, hipReadModeElementType> cudaSourceTextures20;
texture<float, hipTextureType3D, hipReadModeElementType> cudaSourceTextures21;
texture<float, hipTextureType3D, hipReadModeElementType> cudaSourceTextures22;


// material scatter factors
texture<float, hipTextureType1D, hipReadModeElementType> cudaMaterialTextures0;
texture<float, hipTextureType1D, hipReadModeElementType> cudaMaterialTextures1;
texture<float, hipTextureType1D, hipReadModeElementType> cudaMaterialTextures2;

// source ray info
float3 *cudaSourceRayVolOrigin = NULL;
float3 *cudaSourceRayVolDir = NULL;
float *cudaSourceRayTMin = NULL;
float *cudaSourceRayTMax = NULL;
float *cudaSourceScale = NULL; // accounts for source fall off and 1/r^2 attenuation
float *cudaSourceAttenuationCollection = NULL; // output

// detector ray info
float3 *cudaDetectorRayWorldOrigin = NULL;
float3 *cudaDetectorRayWorldDir = NULL;
float3 *cudaDetectorRayVolOrigin = NULL;
float3 *cudaDetectorRayVolDir = NULL;
float *cudaDetectorRayTMin = NULL;
float *cudaDetectorRayTMax = NULL;
float *cudaForwardProjectionCollection = NULL; // output

// info for calculating projection errors
float *cudaBaselineProjection = NULL;
float *cudaForwardProjectionError = NULL;


// info for optimizing which rays to cast
unsigned int *cudaRayIds = NULL;
float *cudaRayPriority = NULL;
float *cudaCurrentForwardProjection = NULL;


__device__ float sampleScatterFactor(int mat, float x) {
  // cell centered to node centered indexing with normalized coordinates
  x = x*((MATERIAL_ANGULAR_SAMPLES-1.0f)/MATERIAL_ANGULAR_SAMPLES) + (0.5f/MATERIAL_ANGULAR_SAMPLES);

  switch (mat) {
  case 0:
    return tex1D(cudaMaterialTextures0, x);
  case 1:
    return tex1D(cudaMaterialTextures1, x);
  case 2:
    return tex1D(cudaMaterialTextures2, x);
  }
  return 0;
}


inline __host__ __device__ float3 operator-(float3 a, int3 b)
{
  return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}
inline __host__ __device__ float3 operator-(int3 a, float3 b)
{
  return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

inline __host__ __device__ float3 operator/(float3 a, int3 b)
{
  return make_float3(a.x / b.x, a.y / b.y, a.z / b.z);
}


__device__ float sampleSourceAttenuation(int combo, float3 x, int3 volDim) {
  // normalize coordinate
  x = x / (volDim-make_int3(1));

  // cell centered to node centered indexing with normalized coordinates
  x = x*((volDim-make_float3(1.0f))/volDim) + (make_float3(0.5f)/volDim);

  switch (combo) {
  case 0:  return tex3D(cudaSourceTextures00, x.x, x.y, x.z);
  case 1:  return tex3D(cudaSourceTextures01, x.x, x.y, x.z);
  case 2:  return tex3D(cudaSourceTextures02, x.x, x.y, x.z);
  case 3:  return tex3D(cudaSourceTextures10, x.x, x.y, x.z);
  case 4:  return tex3D(cudaSourceTextures11, x.x, x.y, x.z);
  case 5:  return tex3D(cudaSourceTextures12, x.x, x.y, x.z);
  case 6:  return tex3D(cudaSourceTextures20, x.x, x.y, x.z);
  case 7:  return tex3D(cudaSourceTextures21, x.x, x.y, x.z);
  case 8:  return tex3D(cudaSourceTextures22, x.x, x.y, x.z);
  }
  return 0; // should never get here
}


__device__ void sampleMaterialsAtPoint(int combo, float3 x, int3 volDim, int3 *mats, float3 *conc) {
  // normalize coordinate
  x = x / (volDim-make_int3(1));

  // cell centered to node centered indexing with normalized coordinates
  x = x*((volDim-make_float3(1.0f))/volDim) + (make_float3(0.5f)/volDim);

  float4 sample;
  switch (combo) {
  case 0:   sample = tex3D(cudaVolumeTextures00, x.x, x.y, x.z);  break;
  case 1:   sample = tex3D(cudaVolumeTextures01, x.x, x.y, x.z);  break;
  case 2:   sample = tex3D(cudaVolumeTextures02, x.x, x.y, x.z);  break;
  case 3:   sample = tex3D(cudaVolumeTextures10, x.x, x.y, x.z);  break;
  case 4:   sample = tex3D(cudaVolumeTextures11, x.x, x.y, x.z);  break;
  case 5:   sample = tex3D(cudaVolumeTextures12, x.x, x.y, x.z);  break;
  case 6:   sample = tex3D(cudaVolumeTextures20, x.x, x.y, x.z);  break;
  case 7:   sample = tex3D(cudaVolumeTextures21, x.x, x.y, x.z);  break;
  case 8:   sample = tex3D(cudaVolumeTextures22, x.x, x.y, x.z);  break;
  }


  // sort the concentrations so highest concentration is at .x, second is at .y
  (*mats) = make_int3(0,1,2);
  (*conc) = make_float3(sample.x, sample.y, sample.z);

  if (conc->x < conc->y) {
    float ti = mats->x;
    mats->x = mats->y;
    mats->y = ti;

    float tf = conc->x;
    conc->x = conc->y;
    conc->y = tf;
  }

  if (conc->y < conc->z) {
    float ti = mats->y;
    mats->y = mats->z;
    mats->z = ti;

    float tf = conc->y;
    conc->y = conc->z;
    conc->z = tf;
  }

  if (conc->x < conc->y) {
    float ti = mats->x;
    mats->x = mats->y;
    mats->y = ti;

    float tf = conc->x;
    conc->x = conc->y;
    conc->y = tf;
  }
}


//==================================================================================================
//==================================================================================================
//==================================================================================================
__global__ void castSourceRays(const float *cudaSourceScale,
                               const float3 *cudaSourceRayVolOrigin,
                               const float3 *cudaSourceRayVolDir,
                               const float *cudaSourceRayTMin,
                               const float *cudaSourceRayTMax,
                               int3 volDim,
                               int collectionSize,
                               float3 matAtten,
                               float3 matDensity,
                               float voxelStepSize,
                               float *cudaSourceAttenuationCollection) {

  unsigned int gx = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned int gy = blockIdx.y*blockDim.y + threadIdx.y;
  unsigned int gz = blockIdx.z*blockDim.z + threadIdx.z;
  if (gx>=volDim.x || gy>=volDim.y || gz>=volDim.z*collectionSize)
    return;
  unsigned int rayIndex = (gz%volDim.z)*volDim.x*volDim.y + gy*volDim.x + gx;
  int combo = gz/volDim.z;


  float3 volOrigin = cudaSourceRayVolOrigin[rayIndex];
  float3 volDir = cudaSourceRayVolDir[rayIndex];
  volDir = volDir / length(volDir);
  float tmin = cudaSourceRayTMin[rayIndex];
  float tmax = cudaSourceRayTMax[rayIndex];

  if (tmin>tmax) {
    // no attenuation from the volume
    cudaSourceAttenuationCollection[combo*volDim.x*volDim.y*volDim.z + rayIndex] = cudaSourceScale[rayIndex];
    return;
  }

  float3 volP0 = volOrigin + tmin*volDir;
  float3 volP1 = volOrigin + tmax*volDir;

  float volLength = length(volP1-volP0);
  int numSteps = volLength / voxelStepSize + 1;


  float lastInterfaceT = tmin;
  int3 lastMat;
  float3 lastConc;

  float3 matLens = make_float3(0,0,0);

  for (int step=0; step<numSteps; step++) {
    float samplef = (step+0.5f) / numSteps;
    float3 volSample = lerp(volP0, volP1, samplef);

    int3 thisMat;
    float3 thisConc;
    sampleMaterialsAtPoint(combo, volSample, volDim, &thisMat, &thisConc);

    // integrate between last step and this one
    if (step > 0) {
      float interfacef;

      // interpolate an interface
      if (thisMat.x != lastMat.x) {
        interfacef = (lastConc.x-lastConc.y) / ((thisConc.x-lastConc.y) - (thisConc.y-lastConc.x));
        interfacef = clamp(interfacef, 0.0f, 1.0f);
        interfacef = lerp((step-0.5f) / numSteps, samplef, interfacef);
      }

      // fixed interface
      else {
        interfacef = (float)step / numSteps;
      }

      float interfaceT = lerp(tmin, tmax, interfacef);

      // add material length before current interface
      float len = interfaceT - lastInterfaceT;
      switch (lastMat.x) {
      case 0:  matLens.x += len;  break;
      case 1:  matLens.y += len;  break;
      case 2:  matLens.z += len;  break;
      }

      lastInterfaceT = interfaceT;
    }

    lastMat = thisMat;
    lastConc = thisConc;
  }

  // add final material length
  float len = tmax - lastInterfaceT;
  switch (lastMat.x) {
  case 0:  matLens.x += len;  break;
  case 1:  matLens.y += len;  break;
  case 2:  matLens.z += len;  break;
  }


  // intergrate attenuations
  cudaSourceAttenuationCollection[combo*volDim.x*volDim.y*volDim.z + rayIndex] =
    cudaSourceScale[rayIndex] * 
    exp(-matLens.x * matDensity.x * matAtten.x +
        -matLens.y * matDensity.y * matAtten.y +
        -matLens.z * matDensity.z * matAtten.z);
}
                               


void MarkovContext::CudaComputeSourceAttenuation(int collectionSize) const {

  int3 volDim = make_int3(mGeometry.GetVolumeNodeSamplesX(),
                          mGeometry.GetVolumeNodeSamplesY(),
                          mGeometry.GetVolumeNodeSamplesZ());

  float3 matAtten = make_float3(mMaterials[0].GetMassAttenuationCoefficient(),
                                mMaterials[1].GetMassAttenuationCoefficient(),
                                mMaterials[2].GetMassAttenuationCoefficient());
  float3 matDensity = make_float3(mMaterials[0].GetDensity(),
                                  mMaterials[1].GetDensity(),
                                  mMaterials[2].GetDensity());
                                

  dim3 dimBlock(8, 8, 1);
  dim3 dimGrid(1+(volDim.x-1) / dimBlock.x, 
               1+(volDim.y-1) / dimBlock.y, 
               1+(collectionSize*volDim.z-1) / dimBlock.z);

  castSourceRays<<<dimGrid, dimBlock>>>(cudaSourceScale,
                                        cudaSourceRayVolOrigin,
                                        cudaSourceRayVolDir,
                                        cudaSourceRayTMin,
                                        cudaSourceRayTMax,
                                        volDim,
                                        collectionSize,
                                        matAtten,
                                        matDensity,
                                        mVoxelStepSize,
                                        cudaSourceAttenuationCollection);


  // bind output to source atten textures
  for (int c=0; c<collectionSize; c++) {
    hipArray **hipArray = NULL;
    switch (c) {
    case 0:
      hipArray = &cudaSourceArray00;
      break;
    case 1:
      hipArray = &cudaSourceArray01;
      break;
    case 2:
      hipArray = &cudaSourceArray02;
      break;
    case 3:
      hipArray = &cudaSourceArray10;
      break;
    case 4:
      hipArray = &cudaSourceArray11;
      break;
    case 5:
      hipArray = &cudaSourceArray12;
      break;
    case 6:
      hipArray = &cudaSourceArray20;
      break;
    case 7:
      hipArray = &cudaSourceArray21;
      break;
    case 8:
      hipArray = &cudaSourceArray22;
      break;
    }


    hipExtent volumeSize = make_hipExtent(mGeometry.GetVolumeNodeSamplesX(),
                                            mGeometry.GetVolumeNodeSamplesY(),
                                            mGeometry.GetVolumeNodeSamplesZ());
    // copy data to 3D array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr((void *)&cudaSourceAttenuationCollection[mGeometry.GetTotalVolumeNodeSamples() * c],
                                              volumeSize.width*sizeof(float), volumeSize.width, volumeSize.height);
    copyParams.dstArray = *hipArray;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyDeviceToDevice;
    checkCudaErrors(hipMemcpy3DAsync(&copyParams));
  }
}


void MarkovContext::CudaGetSourceAttenuation(int collectionSize,
                                             vector< vector<float> > &sourceAttenuationCollection) const {
  // copy results back 
  vector<float> outputData(collectionSize * mGeometry.GetTotalVolumeNodeSamples());
  checkCudaErrors(hipMemcpy(&outputData[0],
                             cudaSourceAttenuationCollection,
                             collectionSize * mGeometry.GetTotalVolumeNodeSamples()*sizeof(float),
                             hipMemcpyDeviceToHost));

  sourceAttenuationCollection.resize(collectionSize);
  for (int c=0; c<collectionSize; c++) {
    sourceAttenuationCollection[c].resize(mCurrentVolumeSourceAttenuation.size());
    for (int i=0; i<mCurrentVolumeSourceAttenuation.size(); i++)
      sourceAttenuationCollection[c][i] = outputData[c*mGeometry.GetTotalVolumeNodeSamples() + i];
  }
}




//==================================================================================================
//==================================================================================================
//==================================================================================================

__global__ void prioritizeDetectorRays(const float3 *cudaDetectorRayWorldOrigin,
                                       const float3 *cudaDetectorRayWorldDir,
                                       const float *cudaDetectorRayTMin,
                                       const float *cudaDetectorRayTMax,
                                       int numRays,
                                       float3 coneOrigin,
                                       float3 coneDir,
                                       float coneCosTheta,
                                       float coneMinDist,
                                       unsigned int *cudaRayIds,
                                       float *cudaRayPriority) {
  unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx >= numRays)
    return;

  float3 rayOrigin = cudaDetectorRayWorldOrigin[idx];
  float3 rayDir = cudaDetectorRayWorldDir[idx];


  float AdD = dot(coneDir, rayDir);
  float cosSqr = (coneCosTheta-COSTHETA_EPS)*(coneCosTheta-COSTHETA_EPS);
  float3 E = rayOrigin - coneOrigin;
  float AdE = dot(coneDir, E);
  float DdE = dot(rayDir, E);
  float EdE = dot(E, E);
  float c2 = AdD*AdD - cosSqr;
  float c1 = AdD*AdE - cosSqr*DdE;
  float c0 = AdE*AdE - cosSqr*EdE;
  float dp;

  float3 point;
  bool hit = false;

  // Solve the quadratic.  Keep only those X for which Dot(A,X-V) >= 0.
  if (fabsf(c2) >= 1e-4) {
    // c2 != 0
    float discr = c1*c1 - c0*c2;
    if (discr > 1e-4) {
      // Q(t) = 0 has two distinct real-valued roots.  However, one or
      // both of them might intersect the portion of the double-sided
      // cone "behind" the vertex.  We are interested only in those
      // intersections "in front" of the vertex.
      float root = sqrtf(discr);
      float invC2 = 1.0f/c2;

      float t = (-c1 - root)*invC2;
      point = rayOrigin + t*rayDir;
      E = point - coneOrigin;
      dp = dot(E, coneDir);
      if (dp > coneMinDist-MINDIST_EPS) {
        hit = true;
      }

      t = (-c1 + root)*invC2;
      point = rayOrigin + t*rayDir;
      E = point - coneOrigin;
      dp = dot(E, coneDir);
      if (dp > coneMinDist-MINDIST_EPS) {
        hit = true;
      }
    }
  }

  //  hit = true;

  if (hit)
    cudaRayPriority[idx] = 1;//(cudaDetectorRayTMax[idx] - cudaDetectorRayTMin[idx]);
  else
    cudaRayPriority[idx] = 0;

  cudaRayIds[idx] = idx;
}


__device__ void castDetectorRay(int rayIndex, int combo,
                                const float3 *cudaDetectorRayWorldOrigin,
                                const float3 *cudaDetectorRayWorldDir,
                                const float3 *cudaDetectorRayVolOrigin,
                                const float3 *cudaDetectorRayVolDir,
                                const float *cudaDetectorRayTMin,
                                const float *cudaDetectorRayTMax,
                                int3 detectorDim,
                                int3 volDim,
                                int collectionSize,
                                float3 matAtten,
                                float3 matDensity,
                                float3 sourcePosition,
                                float voxelStepSize,
                                float *cudaForwardProjectionCollection) {

  float3 worldOrigin = cudaDetectorRayWorldOrigin[rayIndex];
  float3 worldDir = cudaDetectorRayWorldDir[rayIndex];
  float3 volOrigin = cudaDetectorRayVolOrigin[rayIndex];
  float3 volDir = cudaDetectorRayVolDir[rayIndex];
  float tmin = cudaDetectorRayTMin[rayIndex];
  float tmax = cudaDetectorRayTMax[rayIndex];

  if (tmin>tmax) {
    // no attenuation from the volume
    cudaForwardProjectionCollection[combo*detectorDim.x*detectorDim.y*detectorDim.z + rayIndex] = 0;
    return;
  }

  float3 volP0 = volOrigin + tmin*volDir;
  float3 volP1 = volOrigin + tmax*volDir;

  float volLength = length(volP1-volP0);
  int numSteps = volLength / voxelStepSize + 1;


  int3 lastMat;
  float3 lastConc;
  float lastInterfaceT = tmin;

  float forwardProjection = 0;
  float sumDetectorAttenuation = 0;

  float lastInterfaceSourceAtten = sampleSourceAttenuation(combo, volOrigin + tmin*volDir, volDim);


  for (int step=0; step<numSteps; step++) {
    float samplef = (step+0.5f) / numSteps;
    float3 volSample = lerp(volP0, volP1, samplef);

    int3 thisMat;
    float3 thisConc;
    sampleMaterialsAtPoint(combo, volSample, volDim, &thisMat, &thisConc);

    // integrate between last step and this one
    if (step > 0) {
      float interfacef;

      // interpolate an interface
      if (thisMat.x != lastMat.x) {
        interfacef = (lastConc.x-lastConc.y) / ((thisConc.x-lastConc.y) - (thisConc.y-lastConc.x));
        interfacef = clamp(interfacef, 0.0f, 1.0f);
        interfacef = lerp((step-0.5f) / numSteps, samplef, interfacef);
      }

      // fixed interface
      else {
        interfacef = (float)step / numSteps;
      }

      float interfaceT = lerp(tmin, tmax, interfacef);

      //
      // integrate this step
      //
      float tdist = interfaceT - lastInterfaceT;

      // direction to center of material segment from source
      float3 sourceRayDir = normalize((worldOrigin + ((interfaceT+lastInterfaceT)*0.5f)*worldDir) - sourcePosition);
      float ncosScatterAngle = dot(worldDir, sourceRayDir);
      float thisInterfaceSourceAtten = sampleSourceAttenuation(combo, volOrigin + interfaceT*volDir, volDim);

      float density = 0;
      float massAtten = 0;
      switch (lastMat.x) {
      case 0:
        density = matDensity.x;
        massAtten = matAtten.x;
        break;
      case 1:
        density = matDensity.y;
        massAtten = matAtten.y;
        break;
      case 2:
        density = matDensity.z;
        massAtten = matAtten.z;
        break;
      }

      float voxelAttenuation = -tdist * density * massAtten;

      float attenuationFactor;
      if (voxelAttenuation == 0) {
        attenuationFactor = (lastInterfaceSourceAtten+thisInterfaceSourceAtten)*0.5f;
      }
      else {
        double a = lastInterfaceSourceAtten;
        double b = thisInterfaceSourceAtten;
        double c = voxelAttenuation;
        attenuationFactor = (exp(c) * (a+b*(c-1)) - a*(c+1)+b) / (c*c);
      }

      // attenuation between detector and start
      attenuationFactor *= exp(sumDetectorAttenuation);

      forwardProjection += (tdist *
                            sampleScatterFactor(lastMat.x, 0.5f*(1+ncosScatterAngle)) *
                            attenuationFactor);


      sumDetectorAttenuation += voxelAttenuation;

      // store info for next step
      lastInterfaceT = interfaceT;
      lastInterfaceSourceAtten = thisInterfaceSourceAtten;
    }

    lastMat = thisMat;
    lastConc = thisConc;
  }


  //
  // integrate final step
  //
  float interfaceT = tmax;
  float tdist = interfaceT - lastInterfaceT;

  // direction to center of material segment from source
  float3 sourceRayDir = normalize((worldOrigin + ((interfaceT+lastInterfaceT)*0.5f)*worldDir) - sourcePosition);
  float ncosScatterAngle = dot(worldDir, sourceRayDir);
  float thisInterfaceSourceAtten = sampleSourceAttenuation(combo, volOrigin + interfaceT*volDir, volDim);

  float density = 0;
  float massAtten = 0;
  switch (lastMat.x) {
  case 0:
    density = matDensity.x;
    massAtten = matAtten.x;
    break;
  case 1:
    density = matDensity.y;
    massAtten = matAtten.y;
    break;
  case 2:
    density = matDensity.z;
    massAtten = matAtten.z;
    break;
  }

  float voxelAttenuation = -tdist * density * massAtten;

  float attenuationFactor;
  if (voxelAttenuation == 0) {
    attenuationFactor = (lastInterfaceSourceAtten+thisInterfaceSourceAtten)*0.5f;
  }
  else {
    double a = lastInterfaceSourceAtten;
    double b = thisInterfaceSourceAtten;
    double c = voxelAttenuation;
    attenuationFactor = (exp(c) * (a+b*(c-1)) - a*(c+1)+b) / (c*c);
  }

  // attenuation between detector and start
  attenuationFactor *= exp(sumDetectorAttenuation);

  forwardProjection += (tdist *
                        (0.5 * (1+sampleScatterFactor(lastMat.x, ncosScatterAngle))) *
                        attenuationFactor);



  // store result
  cudaForwardProjectionCollection[combo*detectorDim.x*detectorDim.y*detectorDim.z + rayIndex] = forwardProjection;
}
                               

__global__ void castAllDetectorRays(const float3 *cudaDetectorRayWorldOrigin,
                                    const float3 *cudaDetectorRayWorldDir,
                                    const float3 *cudaDetectorRayVolOrigin,
                                    const float3 *cudaDetectorRayVolDir,
                                    const float *cudaDetectorRayTMin,
                                    const float *cudaDetectorRayTMax,
                                    int3 detectorDim,
                                    int3 volDim,
                                    int collectionSize,
                                    float3 matAtten,
                                    float3 matDensity,
                                    float3 sourcePosition,
                                    float voxelStepSize,
                                    float *cudaForwardProjectionCollection) {

  unsigned int gx = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned int gy = blockIdx.y*blockDim.y + threadIdx.y;
  unsigned int gz = blockIdx.z*blockDim.z + threadIdx.z;
  if (gx>=detectorDim.x || gy>=detectorDim.y || gz>=detectorDim.z*collectionSize)
    return;
  unsigned int rayIndex = (gz%detectorDim.z)*detectorDim.x*detectorDim.y + gy*detectorDim.x + gx;
  int combo = gz/detectorDim.z;

  castDetectorRay(rayIndex, combo,
                  cudaDetectorRayWorldOrigin,
                  cudaDetectorRayWorldDir,
                  cudaDetectorRayVolOrigin,
                  cudaDetectorRayVolDir,
                  cudaDetectorRayTMin,
                  cudaDetectorRayTMax,
                  detectorDim,
                  volDim,
                  collectionSize,
                  matAtten,
                  matDensity,
                  sourcePosition,
                  voxelStepSize,
                  cudaForwardProjectionCollection);
}



__global__ void castPrioritizedDetectorRays(const unsigned int *cudaRayIds,
                                            const float *cudaRayPriority,
                                            const float *cudaCurrentForwardProjection,
                                            const float3 *cudaDetectorRayWorldOrigin,
                                            const float3 *cudaDetectorRayWorldDir,
                                            const float3 *cudaDetectorRayVolOrigin,
                                            const float3 *cudaDetectorRayVolDir,
                                            const float *cudaDetectorRayTMin,
                                            const float *cudaDetectorRayTMax,
                                            int3 detectorDim,
                                            int3 volDim,
                                            int collectionSize,
                                            float3 matAtten,
                                            float3 matDensity,
                                            float3 sourcePosition,
                                            float voxelStepSize,
                                            float *cudaForwardProjectionCollection) {

  unsigned int gx = blockIdx.x*blockDim.x + threadIdx.x;
  if (gx>=detectorDim.x*detectorDim.y*detectorDim.z*collectionSize)
    return;

  //unsigned int rayIndexIndex = gx % (detectorDim.x*detectorDim.y*detectorDim.z);
  //int combo = gx / (detectorDim.x*detectorDim.y*detectorDim.z);
  unsigned int rayIndexIndex = gx / collectionSize;
  int combo = gx % collectionSize;

  unsigned int rayIndex = cudaRayIds[rayIndexIndex];
  float priority = cudaRayPriority[rayIndexIndex];

  if (priority == 0) {
    cudaForwardProjectionCollection[combo*
                                    detectorDim.x*
                                    detectorDim.y*
                                    detectorDim.z +
                                    rayIndex] = cudaCurrentForwardProjection[rayIndex];
  }
  else {
    castDetectorRay(rayIndex, combo,
                    cudaDetectorRayWorldOrigin,
                    cudaDetectorRayWorldDir,
                    cudaDetectorRayVolOrigin,
                    cudaDetectorRayVolDir,
                    cudaDetectorRayTMin,
                    cudaDetectorRayTMax,
                    detectorDim,
                    volDim,
                    collectionSize,
                    matAtten,
                    matDensity,
                    sourcePosition,
                    voxelStepSize,
                    cudaForwardProjectionCollection);
  }
}




void MarkovContext::CudaForwardProject(int collectionSize) const {

  int3 volDim = make_int3(mGeometry.GetVolumeNodeSamplesX(),
                          mGeometry.GetVolumeNodeSamplesY(),
                          mGeometry.GetVolumeNodeSamplesZ());
  int3 detectorDim = make_int3(mGeometry.GetDetectorSamplesWidth(),
                               mGeometry.GetDetectorSamplesHeight(),
                               mGeometry.GetNumProjectionAngles());

  float3 matAtten = make_float3(mMaterials[0].GetMassAttenuationCoefficient(),
                                mMaterials[1].GetMassAttenuationCoefficient(),
                                mMaterials[2].GetMassAttenuationCoefficient());
  float3 matDensity = make_float3(mMaterials[0].GetDensity(),
                                  mMaterials[1].GetDensity(),
                                  mMaterials[2].GetDensity());

  float3 sourcePosition = make_float3(mGeometry.GetSourcePosition()[0],
                                      mGeometry.GetSourcePosition()[1],
                                      mGeometry.GetSourcePosition()[2]);
                                

  dim3 dimBlock(8, 8, 1);
  dim3 dimGrid(1+(detectorDim.x-1) / dimBlock.x, 
               1+(detectorDim.y-1) / dimBlock.y, 
               1+(collectionSize*detectorDim.z-1) / dimBlock.z);

  castAllDetectorRays<<<dimGrid, dimBlock>>>(cudaDetectorRayWorldOrigin,
                                             cudaDetectorRayWorldDir,
                                             cudaDetectorRayVolOrigin,
                                             cudaDetectorRayVolDir,
                                             cudaDetectorRayTMin,
                                             cudaDetectorRayTMax,
                                             detectorDim,
                                             volDim,
                                             collectionSize,
                                             matAtten,
                                             matDensity,
                                             sourcePosition,
                                             mVoxelStepSize,
                                             cudaForwardProjectionCollection);
}



void MarkovContext::CudaUpdateForwardProjection(int collectionSize,
                                                const Cone &attenChangeCone) const {

  int3 volDim = make_int3(mGeometry.GetVolumeNodeSamplesX(),
                          mGeometry.GetVolumeNodeSamplesY(),
                          mGeometry.GetVolumeNodeSamplesZ());
  int3 detectorDim = make_int3(mGeometry.GetDetectorSamplesWidth(),
                               mGeometry.GetDetectorSamplesHeight(),
                               mGeometry.GetNumProjectionAngles());

  float3 matAtten = make_float3(mMaterials[0].GetMassAttenuationCoefficient(),
                                mMaterials[1].GetMassAttenuationCoefficient(),
                                mMaterials[2].GetMassAttenuationCoefficient());
  float3 matDensity = make_float3(mMaterials[0].GetDensity(),
                                  mMaterials[1].GetDensity(),
                                  mMaterials[2].GetDensity());

  float3 sourcePosition = make_float3(mGeometry.GetSourcePosition()[0],
                                      mGeometry.GetSourcePosition()[1],
                                      mGeometry.GetSourcePosition()[2]);

  // prioritize each ray
  dim3 dimBlockPrioritize(32,1,1);
  dim3 dimGridPrioritize(1+(mGeometry.GetTotalProjectionSamples()-1) / dimBlockPrioritize.x, 1, 1);
  prioritizeDetectorRays<<<dimGridPrioritize, dimBlockPrioritize>>>(cudaDetectorRayWorldOrigin,
                                                                    cudaDetectorRayWorldDir,
                                                                    cudaDetectorRayTMin,
                                                                    cudaDetectorRayTMax,
                                                                    mGeometry.GetTotalProjectionSamples(),
                                                                    make_float3(attenChangeCone.mOrigin[0],
                                                                                attenChangeCone.mOrigin[1],
                                                                                attenChangeCone.mOrigin[2]),
                                                                    make_float3(attenChangeCone.mDir[0],
                                                                                attenChangeCone.mDir[1],
                                                                                attenChangeCone.mDir[2]),
                                                                    attenChangeCone.mCosTheta,
                                                                    attenChangeCone.mMinDist,
                                                                    cudaRayIds,
                                                                    cudaRayPriority);


  // sort rays by priority
  thrust::device_ptr<unsigned int> thrustIds = thrust::device_pointer_cast(cudaRayIds);
  thrust::device_ptr<float> thrustPriorities = thrust::device_pointer_cast(cudaRayPriority);
  thrust::sort_by_key(thrustPriorities, thrustPriorities+mGeometry.GetTotalProjectionSamples(), thrustIds);
  //thrust::stable_sort_by_key(thrustPriorities, thrustPriorities+mGeometry.GetTotalProjectionSamples(), thrustIds);

  /*
  vector<float> priorities(mGeometry.GetTotalProjectionSamples());
  checkCudaErrors(hipMemcpy(&priorities[0], cudaRayPriority, sizeof(float)*mGeometry.GetTotalProjectionSamples(), hipMemcpyDeviceToHost));

  int skippedRays = 0;
  for (int i=0; i<priorities.size(); i++) {
    if (priorities[i] == 0) {
      skippedRays++;
    }
  }

  std::cerr<<"skipped "<<skippedRays<<" of "<<priorities.size()<<std::endl;
  */


  // cast only non-zero priority rays
  /*
  dim3 dimBlock(8, 8, 1);
  dim3 dimGrid(1+(detectorDim.x-1) / dimBlock.x, 
               1+(detectorDim.y-1) / dimBlock.y, 
               1+(collectionSize*detectorDim.z-1) / dimBlock.z);
  */

  dim3 dimBlock(32,1,1);
  dim3 dimGrid(1+(collectionSize * mGeometry.GetTotalProjectionSamples()-1) / dimBlockPrioritize.x, 1, 1);


  castPrioritizedDetectorRays<<<dimGrid, dimBlock>>>(cudaRayIds,
                                                     cudaRayPriority,
                                                     cudaCurrentForwardProjection,
                                                     cudaDetectorRayWorldOrigin,
                                                     cudaDetectorRayWorldDir,
                                                     cudaDetectorRayVolOrigin,
                                                     cudaDetectorRayVolDir,
                                                     cudaDetectorRayTMin,
                                                     cudaDetectorRayTMax,
                                                     detectorDim,
                                                     volDim,
                                                     collectionSize,
                                                     matAtten,
                                                     matDensity,
                                                     sourcePosition,
                                                     mVoxelStepSize,
                                                     cudaForwardProjectionCollection);
}


void MarkovContext::CudaGetForwardProjection(int collectionSize,
                                             vector< vector<float> > &forwardProjectionCollection) const {
  // copy results back
  vector<float> outputData(collectionSize * mGeometry.GetTotalProjectionSamples());
  checkCudaErrors(hipMemcpy(&outputData[0],
                             cudaForwardProjectionCollection,
                             collectionSize * mGeometry.GetTotalProjectionSamples()*sizeof(float),
                             hipMemcpyDeviceToHost));

  forwardProjectionCollection.resize(collectionSize);
  for (int c=0; c<collectionSize; c++) {
    forwardProjectionCollection[c].resize(mGeometry.GetTotalProjectionSamples());
    for (int i=0; i<mGeometry.GetTotalProjectionSamples(); i++)
      forwardProjectionCollection[c][i] = outputData[c*mGeometry.GetTotalProjectionSamples() + i];
  }
}



//==================================================================================================
//==================================================================================================
//==================================================================================================
__global__ void projectionToError(int totalProjectionSamples,
                                  int collectionSize,
                                  const float *baselineProjection,
                                  const float *forwardProjection,
                                  float *forwardProjectionError) {

  unsigned int cpi = blockIdx.x*blockDim.x + threadIdx.x;
  if (cpi >= totalProjectionSamples*collectionSize)
    return;

  int pi = cpi % totalProjectionSamples;
  float df = forwardProjection[cpi] - baselineProjection[pi];
  forwardProjectionError[cpi] = df*df;
}


void MarkovContext::CudaGetProjectionError(int collectionSize, vector<float> &errors) const {

  // compute squared errors
  int totalProjectionSamples = mGeometry.GetTotalProjectionSamples();
  dim3 dimBlock(32, 1, 1);
  dim3 dimGrid(1+((totalProjectionSamples*collectionSize)-1) / dimBlock.x, 1, 1);

  projectionToError<<<dimGrid, dimBlock>>>(totalProjectionSamples,
                                           collectionSize,
                                           cudaBaselineProjection,
                                           cudaForwardProjectionCollection,
                                           cudaForwardProjectionError);

  // use thrust to sum the errors for each material combo
  thrust::device_ptr<float> dev_ptr = thrust::device_pointer_cast(cudaForwardProjectionError);
  errors.resize(collectionSize);
  for (int c=0; c<collectionSize; c++) {
    errors[c] = (thrust::reduce(dev_ptr+c*totalProjectionSamples,
                                dev_ptr+(c+1)*totalProjectionSamples)
                 * mGeometry.GetDetectorPixelArea());
  }
}



//==================================================================================================
//==================================================================================================
//==================================================================================================
__global__ void cudaUpdateVolume1(float4 *vol,
                                  int idx,
                                  int mat) {

  // only a single thread needs to do anything
  if (blockIdx.x != 0 || threadIdx.x != 0)
    return;

  float4 nv = make_float4(0,0,0,0);
  switch (mat) {
  case 0:  nv.x = 1;  break;
  case 1:  nv.y = 1;  break;
  case 2:  nv.z = 1;  break;
  }

  vol[idx] = nv;
}


__global__ void cudaUpdateVolume2(float4 *vol,
                                  int idx,
                                  int mat,
                                  int idx2,
                                  int mat2) {

  // only a single thread needs to do anything
  if (blockIdx.x != 0 || threadIdx.x != 0)
    return;

  float4 nv = make_float4(0,0,0,0);
  switch (mat) {
  case 0:  nv.x = 1;  break;
  case 1:  nv.y = 1;  break;
  case 2:  nv.z = 1;  break;
  }
  vol[idx] = nv;

  float4 nv2 = make_float4(0,0,0,0);
  switch (mat2) {
  case 0:  nv2.x = 1;  break;
  case 1:  nv2.y = 1;  break;
  case 2:  nv2.z = 1;  break;
  }
  vol[idx2] = nv2;
}


void MarkovContext::CudaAcceptNextConfig(int c) const {
  // set the accepted volume as current
  checkCudaErrors(hipMemcpyAsync(cudaVolumeLinearCurrent,
                                  cudaVolumeLinearCollection + c*mGeometry.GetTotalVolumeNodeSamples(),
                                  sizeof(float4)*mGeometry.GetTotalVolumeNodeSamples(),
                                  hipMemcpyDeviceToDevice));

  // set the accepted forward projection as current
  checkCudaErrors(hipMemcpyAsync(cudaCurrentForwardProjection,
                                  cudaForwardProjectionCollection + c*mGeometry.GetTotalProjectionSamples(),
                                  sizeof(float)*mGeometry.GetTotalProjectionSamples(),
                                  hipMemcpyDeviceToDevice));
}


void MarkovContext::CudaSetCurrentVolume(const vector<unsigned char> &matids) const {

  // setup individual channels per material
  vector<float4> volumeData(mGeometry.GetTotalVolumeNodeSamples());
  memset(&volumeData[0], 0, sizeof(float4)*mGeometry.GetTotalVolumeNodeSamples());
  for (int i=0; i<mGeometry.GetTotalVolumeNodeSamples(); i++) {
    switch (matids[i]) {
    case 0:  volumeData[i].x = 1;  break;
    case 1:  volumeData[i].y = 1;  break;
    case 2:  volumeData[i].z = 1;  break;
    }
  }

  checkCudaErrors(hipMemcpy(cudaVolumeLinearCurrent,
                             &volumeData[0],
                             sizeof(float4)*mGeometry.GetTotalVolumeNodeSamples(),
                             hipMemcpyHostToDevice));
}


void MarkovContext::CudaGetVolumeCollection(vector< vector<unsigned char> > &volumeCollection) const {
  volumeCollection.resize(NUM_MATERIALS*NUM_MATERIALS);
  for (int c=0; c<NUM_MATERIALS*NUM_MATERIALS; c++) {

    vector<float4> fvol(mGeometry.GetTotalVolumeNodeSamples());

    checkCudaErrors(hipMemcpy(&fvol[0],
                               cudaVolumeLinearCollection + c*mGeometry.GetTotalVolumeNodeSamples(),
                               sizeof(float4)*mGeometry.GetTotalVolumeNodeSamples(),
                               hipMemcpyDeviceToHost));


    volumeCollection[c].resize(mGeometry.GetTotalVolumeNodeSamples());
    for (int i=0; i<mGeometry.GetTotalVolumeNodeSamples(); i++) {
      if (fvol[i].x == 1)
        volumeCollection[c][i] = 0;
      else if (fvol[i].y == 1)
        volumeCollection[c][i] = 1;
      else if (fvol[i].z == 1)
        volumeCollection[c][i] = 2;
      else {
        std::cerr<<"bogus volume!"<<std::endl;
        exit(0);
      }
    }
    
  }
}


void MarkovContext::CudaSetVolumeCollection(const GibbsProposal &proposal) const {

  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
  hipExtent volumeSize = make_hipExtent(mGeometry.GetVolumeNodeSamplesX(),
                                          mGeometry.GetVolumeNodeSamplesY(),
                                          mGeometry.GetVolumeNodeSamplesZ());

  for (int c=0; c<NUM_MATERIALS*NUM_MATERIALS; c++) {

    float4 *cudaVolume = cudaVolumeLinearCollection + c*mGeometry.GetTotalVolumeNodeSamples();

    // copy current volume to collection volume
    checkCudaErrors(hipMemcpyAsync(cudaVolume,
                                    cudaVolumeLinearCurrent,
                                    sizeof(float4)*mGeometry.GetTotalVolumeNodeSamples(),
                                    hipMemcpyDeviceToDevice));
  }

  for (int c=0; c<NUM_MATERIALS*NUM_MATERIALS; c++) {
    float4 *cudaVolume = cudaVolumeLinearCollection + c*mGeometry.GetTotalVolumeNodeSamples();

    // apply changes
    // first proposal only
    if (proposal.first>=0 && proposal.second<0) {
      dim3 dimBlock(1,1,1);
      dim3 dimGrid(1,1,1);
      cudaUpdateVolume1<<<dimGrid, dimBlock>>>(cudaVolume,
                                               proposal.first,
                                               c%NUM_MATERIALS);
    }

    // apply changes
    // both proposals
    else if (proposal.first>=0 && proposal.second>=0) {
      dim3 dimBlock(1,1,1);
      dim3 dimGrid(1,1,1);
      cudaUpdateVolume2<<<dimGrid, dimBlock>>>(cudaVolume,
                                               proposal.first,
                                               c%NUM_MATERIALS,
                                               proposal.second,
                                               c/NUM_MATERIALS);
    }
  }

    
  for (int c=0; c<NUM_MATERIALS*NUM_MATERIALS; c++) {
    float4 *cudaVolume = cudaVolumeLinearCollection + c*mGeometry.GetTotalVolumeNodeSamples();

    hipArray **hipArray = NULL;
    switch (c) {
    case 0:
      hipArray = &cudaVolumeArray00;
      break;
    case 1:
      hipArray = &cudaVolumeArray01;
      break;
    case 2:
      hipArray = &cudaVolumeArray02;
      break;
    case 3:
      hipArray = &cudaVolumeArray10;
      break;
    case 4:
      hipArray = &cudaVolumeArray11;
      break;
    case 5:
      hipArray = &cudaVolumeArray12;
      break;
    case 6:
      hipArray = &cudaVolumeArray20;
      break;
    case 7:
      hipArray = &cudaVolumeArray21;
      break;
    case 8:
      hipArray = &cudaVolumeArray22;
      break;
    }

    // copy data to 3D array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr(cudaVolume, volumeSize.width*sizeof(float4), volumeSize.width, volumeSize.height);
    copyParams.dstArray = *hipArray;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyDeviceToDevice;
    checkCudaErrors(hipMemcpy3DAsync(&copyParams));
  }
}



//==================================================================================================
//==================================================================================================
//==================================================================================================
void MarkovContext::CudaSetBaselineProjection() const {
  checkCudaErrors(hipMalloc(&cudaBaselineProjection, sizeof(float)*mBaselineProjection.size()));
  checkCudaErrors(hipMemcpy(cudaBaselineProjection, &mBaselineProjection[0], sizeof(float)*mBaselineProjection.size(), hipMemcpyHostToDevice));
}


template <typename T>
void SetTextureParams(T *cudaTexture) {
  cudaTexture->normalized = true;
  cudaTexture->filterMode = hipFilterModeLinear;
  cudaTexture->addressMode[0] = hipAddressModeClamp;
  cudaTexture->addressMode[1] = hipAddressModeClamp;
  cudaTexture->addressMode[2] = hipAddressModeClamp;
}

void MarkovContext::CudaInitialize() {

  // This will pick the best possible CUDA capable device
  int devID = findCudaDevice(0, NULL);

  // get device name
  hipDeviceProp_t deviceProps;
  checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
  printf("CUDA device [%s]\n", deviceProps.name);


  // initialize material data
  for (int m=0; m<mMaterials.size(); m++) {
    vector<float> scatterFactors;
    mMaterials[m].GetScatterFactorArray(scatterFactors);

    // Allocate array and copy image data
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray *cuArray;
    checkCudaErrors(hipMallocArray(&cuArray,
                                    &channelDesc,
                                    (int)scatterFactors.size(),
                                    1,
                                    hipArrayDefault));
    checkCudaErrors(hipMemcpyToArray(cuArray,
                                      0,
                                      0,
                                      &scatterFactors[0],
                                      (int)scatterFactors.size() * sizeof(float),
                                      hipMemcpyHostToDevice));


    texture<float, 1, hipReadModeElementType> *cudaMaterialTexture = NULL;
    switch (m) {
    case 0:
      cudaMaterialTexture = &cudaMaterialTextures0; break;
    case 1:
      cudaMaterialTexture = &cudaMaterialTextures1; break;
    case 2:
      cudaMaterialTexture = &cudaMaterialTextures2; break;
    }

    // Set texture parameters
    cudaMaterialTexture->addressMode[0] = hipAddressModeClamp;
    cudaMaterialTexture->addressMode[1] = hipAddressModeClamp;
    cudaMaterialTexture->filterMode = hipFilterModeLinear;
    cudaMaterialTexture->normalized = true;    // access with normalized texture coordinates
    
    // Bind the array to the texture
    checkCudaErrors(hipBindTextureToArray(*cudaMaterialTexture, cuArray, channelDesc));
  }


  // volume data
  hipExtent volumeSize = make_hipExtent(mGeometry.GetVolumeNodeSamplesX(),
                                          mGeometry.GetVolumeNodeSamplesY(),
                                          mGeometry.GetVolumeNodeSamplesZ());
  checkCudaErrors(hipMalloc(&cudaVolumeLinearCurrent, sizeof(float4)*mGeometry.GetTotalVolumeNodeSamples()));
  checkCudaErrors(hipMalloc(&cudaVolumeLinearCollection, sizeof(float4)*mGeometry.GetTotalVolumeNodeSamples()*NUM_MATERIALS*NUM_MATERIALS));
  hipChannelFormatDesc channelDesc4 = hipCreateChannelDesc<float4>();
  checkCudaErrors(hipMalloc3DArray(&cudaVolumeArray00, &channelDesc4, volumeSize));
  checkCudaErrors(hipMalloc3DArray(&cudaVolumeArray01, &channelDesc4, volumeSize));
  checkCudaErrors(hipMalloc3DArray(&cudaVolumeArray02, &channelDesc4, volumeSize));
  checkCudaErrors(hipMalloc3DArray(&cudaVolumeArray10, &channelDesc4, volumeSize));
  checkCudaErrors(hipMalloc3DArray(&cudaVolumeArray11, &channelDesc4, volumeSize));
  checkCudaErrors(hipMalloc3DArray(&cudaVolumeArray12, &channelDesc4, volumeSize));
  checkCudaErrors(hipMalloc3DArray(&cudaVolumeArray20, &channelDesc4, volumeSize));
  checkCudaErrors(hipMalloc3DArray(&cudaVolumeArray21, &channelDesc4, volumeSize));
  checkCudaErrors(hipMalloc3DArray(&cudaVolumeArray22, &channelDesc4, volumeSize));

  SetTextureParams(&cudaVolumeTextures00);
  SetTextureParams(&cudaVolumeTextures01);
  SetTextureParams(&cudaVolumeTextures02);
  SetTextureParams(&cudaVolumeTextures10);
  SetTextureParams(&cudaVolumeTextures11);
  SetTextureParams(&cudaVolumeTextures12);
  SetTextureParams(&cudaVolumeTextures20);
  SetTextureParams(&cudaVolumeTextures21);
  SetTextureParams(&cudaVolumeTextures22);
  checkCudaErrors(hipBindTextureToArray(cudaVolumeTextures00, cudaVolumeArray00, channelDesc4));
  checkCudaErrors(hipBindTextureToArray(cudaVolumeTextures01, cudaVolumeArray01, channelDesc4));
  checkCudaErrors(hipBindTextureToArray(cudaVolumeTextures02, cudaVolumeArray02, channelDesc4));
  checkCudaErrors(hipBindTextureToArray(cudaVolumeTextures10, cudaVolumeArray10, channelDesc4));
  checkCudaErrors(hipBindTextureToArray(cudaVolumeTextures11, cudaVolumeArray11, channelDesc4));
  checkCudaErrors(hipBindTextureToArray(cudaVolumeTextures12, cudaVolumeArray12, channelDesc4));
  checkCudaErrors(hipBindTextureToArray(cudaVolumeTextures20, cudaVolumeArray20, channelDesc4));
  checkCudaErrors(hipBindTextureToArray(cudaVolumeTextures21, cudaVolumeArray21, channelDesc4));
  checkCudaErrors(hipBindTextureToArray(cudaVolumeTextures22, cudaVolumeArray22, channelDesc4));


  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
  checkCudaErrors(hipMalloc3DArray(&cudaSourceArray00, &channelDesc, volumeSize));
  checkCudaErrors(hipMalloc3DArray(&cudaSourceArray01, &channelDesc, volumeSize));
  checkCudaErrors(hipMalloc3DArray(&cudaSourceArray02, &channelDesc, volumeSize));
  checkCudaErrors(hipMalloc3DArray(&cudaSourceArray10, &channelDesc, volumeSize));
  checkCudaErrors(hipMalloc3DArray(&cudaSourceArray11, &channelDesc, volumeSize));
  checkCudaErrors(hipMalloc3DArray(&cudaSourceArray12, &channelDesc, volumeSize));
  checkCudaErrors(hipMalloc3DArray(&cudaSourceArray20, &channelDesc, volumeSize));
  checkCudaErrors(hipMalloc3DArray(&cudaSourceArray21, &channelDesc, volumeSize));
  checkCudaErrors(hipMalloc3DArray(&cudaSourceArray22, &channelDesc, volumeSize));

  SetTextureParams(&cudaSourceTextures00);
  SetTextureParams(&cudaSourceTextures01);
  SetTextureParams(&cudaSourceTextures02);
  SetTextureParams(&cudaSourceTextures10);
  SetTextureParams(&cudaSourceTextures11);
  SetTextureParams(&cudaSourceTextures12);
  SetTextureParams(&cudaSourceTextures20);
  SetTextureParams(&cudaSourceTextures21);
  SetTextureParams(&cudaSourceTextures22);
  checkCudaErrors(hipBindTextureToArray(cudaSourceTextures00, cudaSourceArray00, channelDesc));
  checkCudaErrors(hipBindTextureToArray(cudaSourceTextures01, cudaSourceArray01, channelDesc));
  checkCudaErrors(hipBindTextureToArray(cudaSourceTextures02, cudaSourceArray02, channelDesc));
  checkCudaErrors(hipBindTextureToArray(cudaSourceTextures10, cudaSourceArray10, channelDesc));
  checkCudaErrors(hipBindTextureToArray(cudaSourceTextures11, cudaSourceArray11, channelDesc));
  checkCudaErrors(hipBindTextureToArray(cudaSourceTextures12, cudaSourceArray12, channelDesc));
  checkCudaErrors(hipBindTextureToArray(cudaSourceTextures20, cudaSourceArray20, channelDesc));
  checkCudaErrors(hipBindTextureToArray(cudaSourceTextures21, cudaSourceArray21, channelDesc));
  checkCudaErrors(hipBindTextureToArray(cudaSourceTextures22, cudaSourceArray22, channelDesc));


  // upload all of the ray info
  checkCudaErrors(hipMalloc(&cudaSourceRayVolOrigin, sizeof(float3)*mSourceRayVolOrigin.size()));
  checkCudaErrors(hipMemcpy(cudaSourceRayVolOrigin, &mSourceRayVolOrigin[0], sizeof(float3)*mSourceRayVolOrigin.size(), hipMemcpyHostToDevice));
  checkCudaErrors(hipMalloc(&cudaSourceRayVolDir, sizeof(float3)*mSourceRayVolDir.size()));
  checkCudaErrors(hipMemcpy(cudaSourceRayVolDir, &mSourceRayVolDir[0], sizeof(float3)*mSourceRayVolDir.size(), hipMemcpyHostToDevice));
  checkCudaErrors(hipMalloc(&cudaSourceRayTMin, sizeof(float)*mSourceRayTMin.size()));
  checkCudaErrors(hipMemcpy(cudaSourceRayTMin, &mSourceRayTMin[0], sizeof(float)*mSourceRayTMin.size(), hipMemcpyHostToDevice));
  checkCudaErrors(hipMalloc(&cudaSourceRayTMax, sizeof(float)*mSourceRayTMax.size()));
  checkCudaErrors(hipMemcpy(cudaSourceRayTMax, &mSourceRayTMax[0], sizeof(float)*mSourceRayTMax.size(), hipMemcpyHostToDevice));
  checkCudaErrors(hipMalloc(&cudaDetectorRayVolOrigin, sizeof(float3)*mDetectorRayVolOrigin.size()));
  checkCudaErrors(hipMemcpy(cudaDetectorRayVolOrigin, &mDetectorRayVolOrigin[0], sizeof(float3)*mDetectorRayVolOrigin.size(), hipMemcpyHostToDevice));
  checkCudaErrors(hipMalloc(&cudaDetectorRayVolDir, sizeof(float3)*mDetectorRayVolDir.size()));
  checkCudaErrors(hipMemcpy(cudaDetectorRayVolDir, &mDetectorRayVolDir[0], sizeof(float3)*mDetectorRayVolDir.size(), hipMemcpyHostToDevice));
  checkCudaErrors(hipMalloc(&cudaDetectorRayWorldOrigin, sizeof(float3)*mDetectorRayOrigin.size()));
  checkCudaErrors(hipMemcpy(cudaDetectorRayWorldOrigin, &mDetectorRayOrigin[0], sizeof(float3)*mDetectorRayOrigin.size(), hipMemcpyHostToDevice));
  checkCudaErrors(hipMalloc(&cudaDetectorRayWorldDir, sizeof(float3)*mDetectorRayDir.size()));
  checkCudaErrors(hipMemcpy(cudaDetectorRayWorldDir, &mDetectorRayDir[0], sizeof(float3)*mDetectorRayDir.size(), hipMemcpyHostToDevice));
  checkCudaErrors(hipMalloc(&cudaDetectorRayTMin, sizeof(float)*mDetectorRayTMin.size()));
  checkCudaErrors(hipMemcpy(cudaDetectorRayTMin, &mDetectorRayTMin[0], sizeof(float)*mDetectorRayTMin.size(), hipMemcpyHostToDevice));
  checkCudaErrors(hipMalloc(&cudaDetectorRayTMax, sizeof(float)*mDetectorRayTMax.size()));
  checkCudaErrors(hipMemcpy(cudaDetectorRayTMax, &mDetectorRayTMax[0], sizeof(float)*mDetectorRayTMax.size(), hipMemcpyHostToDevice));

  checkCudaErrors(hipMalloc(&cudaRayIds, sizeof(unsigned int)*mGeometry.GetTotalProjectionSamples()));
  checkCudaErrors(hipMalloc(&cudaRayPriority, sizeof(float)*mGeometry.GetTotalProjectionSamples()));
  checkCudaErrors(hipMalloc(&cudaCurrentForwardProjection, sizeof(float)*mGeometry.GetTotalProjectionSamples()));


  // precompute some source attenuation info
  vector<float> sourceScale(mGeometry.GetTotalVolumeNodeSamples());
  for (int nvi=0; nvi<mGeometry.GetTotalVolumeNodeSamples(); nvi++) {
    int x,y,z;
    mGeometry.VolumeIndexToNodeCoord(nvi, x,y,z);

    Vec3f voxelPosition;
    mGeometry.VolumeToWorld(Vec3f((float)x,(float)y,(float)z), voxelPosition);

    Vec3f diff = voxelPosition - mGeometry.GetSourcePosition();
    float maxt = diff.Length();
    Vec3f dir = diff / maxt;
    
    sourceScale[nvi] = mGeometry.GetSourceIntensityThroughPoint(voxelPosition) / (maxt*maxt);
  }

  checkCudaErrors(hipMalloc(&cudaSourceScale, sizeof(float)*sourceScale.size()));
  checkCudaErrors(hipMemcpy(cudaSourceScale, &sourceScale[0], sizeof(float)*sourceScale.size(), hipMemcpyHostToDevice));


  checkCudaErrors(hipMalloc(&cudaSourceAttenuationCollection, sizeof(float)*mGeometry.GetTotalVolumeNodeSamples() * NUM_MATERIALS*NUM_MATERIALS));
  checkCudaErrors(hipMalloc(&cudaForwardProjectionCollection, sizeof(float)*mGeometry.GetTotalProjectionSamples() * NUM_MATERIALS*NUM_MATERIALS));
  checkCudaErrors(hipMalloc(&cudaForwardProjectionError, sizeof(float)*mGeometry.GetTotalProjectionSamples() * NUM_MATERIALS*NUM_MATERIALS));

}


void MarkovContext::CudaShutdown() const {
  hipDeviceReset();
}
