#include "hip/hip_runtime.h"

#ifdef _WIN32
#define WIN32_LEAN_AND_MEAN
#endif

#define MAX_GPUS 2


#include <Application/BackscatterReconstruction/Algorithm/markov.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>

#include <Application/BackscatterReconstruction/Algorithm/cuda_common/helper_functions.h>
#include <Application/BackscatterReconstruction/Algorithm/cuda_common/hip/hip_runtime_api.h>
#include <Application/BackscatterReconstruction/Algorithm/cuda_common/cutil_math.h>

//#define CUDA_ENABLE_UPDATE_FORWARD_PROJECTION


#ifdef WIN32
#include <windows.h>
class Timer {
  public:
  Timer(const char *name) :
    mName(name) {
    mTotalTime.QuadPart = 0;
    mStartTime.QuadPart = 0;
  }

  ~Timer() {
    LARGE_INTEGER frequency;
    QueryPerformanceFrequency(&frequency);
    std::cerr<<mName<<": "<<(double)mTotalTime.QuadPart / frequency.QuadPart<<std::endl;
  }


  void Start() {
    QueryPerformanceCounter(&mStartTime);
  }

  void Stop() {
    LARGE_INTEGER stopTime;
    QueryPerformanceCounter(&stopTime);
    mTotalTime.QuadPart += stopTime.QuadPart - mStartTime.QuadPart;
  }


  const char *mName;
  LARGE_INTEGER mTotalTime;
  LARGE_INTEGER mStartTime;
};

#if 0
#define TIMER_START(name)   static Timer timer(name);  timer.Start();
#define TIMER_STOP                              \
  for (int g=0; g<mGpuIds.size(); g++) {        \
    checkCudaErrors(hipSetDevice(mGpuIds[g])); \
    checkCudaErrors(hipDeviceSynchronize());   \
  }                                             \
  timer.Stop();
#else
#define TIMER_START(name)
#define TIMER_STOP
#endif

#else

#define TIMER_START(name)
#define TIMER_STOP

#endif



// all the memory / texture references needed for each gpu.  Texture references 
// are magical and automatically get duplicated for each gpu.

// material volumes in the collection
float4 *cudaVolumeLinearCollection[MAX_GPUS] = { NULL, NULL };
hipArray *cudaVolumeArray00[MAX_GPUS] = { NULL, NULL };
hipArray *cudaVolumeArray01[MAX_GPUS] = { NULL, NULL };
hipArray *cudaVolumeArray02[MAX_GPUS] = { NULL, NULL };
hipArray *cudaVolumeArray10[MAX_GPUS] = { NULL, NULL };
hipArray *cudaVolumeArray11[MAX_GPUS] = { NULL, NULL };
hipArray *cudaVolumeArray12[MAX_GPUS] = { NULL, NULL };
hipArray *cudaVolumeArray20[MAX_GPUS] = { NULL, NULL };
hipArray *cudaVolumeArray21[MAX_GPUS] = { NULL, NULL };
hipArray *cudaVolumeArray22[MAX_GPUS] = { NULL, NULL };
texture<float4, hipTextureType3D, hipReadModeElementType> cudaVolumeTextures00;
texture<float4, hipTextureType3D, hipReadModeElementType> cudaVolumeTextures01;
texture<float4, hipTextureType3D, hipReadModeElementType> cudaVolumeTextures02;
texture<float4, hipTextureType3D, hipReadModeElementType> cudaVolumeTextures10;
texture<float4, hipTextureType3D, hipReadModeElementType> cudaVolumeTextures11;
texture<float4, hipTextureType3D, hipReadModeElementType> cudaVolumeTextures12;
texture<float4, hipTextureType3D, hipReadModeElementType> cudaVolumeTextures20;
texture<float4, hipTextureType3D, hipReadModeElementType> cudaVolumeTextures21;
texture<float4, hipTextureType3D, hipReadModeElementType> cudaVolumeTextures22;


// source attenuation volumes in the collection
hipArray *cudaSourceArray00[MAX_GPUS] = { NULL, NULL };
hipArray *cudaSourceArray01[MAX_GPUS] = { NULL, NULL };
hipArray *cudaSourceArray02[MAX_GPUS] = { NULL, NULL };
hipArray *cudaSourceArray10[MAX_GPUS] = { NULL, NULL };
hipArray *cudaSourceArray11[MAX_GPUS] = { NULL, NULL };
hipArray *cudaSourceArray12[MAX_GPUS] = { NULL, NULL };
hipArray *cudaSourceArray20[MAX_GPUS] = { NULL, NULL };
hipArray *cudaSourceArray21[MAX_GPUS] = { NULL, NULL };
hipArray *cudaSourceArray22[MAX_GPUS] = { NULL, NULL };
texture<float, hipTextureType3D, hipReadModeElementType> cudaSourceTextures00;
texture<float, hipTextureType3D, hipReadModeElementType> cudaSourceTextures01;
texture<float, hipTextureType3D, hipReadModeElementType> cudaSourceTextures02;
texture<float, hipTextureType3D, hipReadModeElementType> cudaSourceTextures10;
texture<float, hipTextureType3D, hipReadModeElementType> cudaSourceTextures11;
texture<float, hipTextureType3D, hipReadModeElementType> cudaSourceTextures12;
texture<float, hipTextureType3D, hipReadModeElementType> cudaSourceTextures20;
texture<float, hipTextureType3D, hipReadModeElementType> cudaSourceTextures21;
texture<float, hipTextureType3D, hipReadModeElementType> cudaSourceTextures22;


// material scatter factors
texture<float, hipTextureType1D, hipReadModeElementType> cudaMaterialTextures0;
texture<float, hipTextureType1D, hipReadModeElementType> cudaMaterialTextures1;
texture<float, hipTextureType1D, hipReadModeElementType> cudaMaterialTextures2;
hipArray* cudaMaterialArray[MAX_GPUS][NUM_MATERIALS];


// source ray info
float3 *cudaSourceRayVolOrigin[MAX_GPUS] = { NULL, NULL };
float3 *cudaSourceRayVolDir[MAX_GPUS] = { NULL, NULL };
float *cudaSourceRayTMin[MAX_GPUS] = { NULL, NULL };
float *cudaSourceRayTMax[MAX_GPUS] = { NULL, NULL };
float *cudaSourceScale[MAX_GPUS] = { NULL, NULL }; // accounts for source 1/r^2 attenuation
float *cudaSourceAttenuationCollection[MAX_GPUS] = { NULL, NULL }; // output
texture<float, hipTextureType2D, hipReadModeElementType> cudaSourceFalloffTexture;
hipArray *cudaFalloffArray[MAX_GPUS] = { NULL, NULL };
float *cudaProjectionAngles[MAX_GPUS] = { NULL, NULL };

// detector ray info
float3 *cudaDetectorRayWorldOrigin[MAX_GPUS] = { NULL, NULL };
float3 *cudaDetectorRayWorldDir[MAX_GPUS] = { NULL, NULL };
float3 *cudaDetectorRayVolOrigin[MAX_GPUS] = { NULL, NULL };
float3 *cudaDetectorRayVolDir[MAX_GPUS] = { NULL, NULL };
float *cudaDetectorRayTMin[MAX_GPUS] = { NULL, NULL };
float *cudaDetectorRayTMax[MAX_GPUS] = { NULL, NULL };
float *cudaForwardProjectionCollection[MAX_GPUS] = { NULL, NULL }; // output

// info for calculating projection errors
float *cudaBaselineProjection[MAX_GPUS] = { NULL, NULL };
float *cudaForwardProjectionError[MAX_GPUS] = { NULL, NULL };


// info for optimizing which rays to cast
unsigned int *cudaRayIds[MAX_GPUS] = { NULL, NULL };
float *cudaRayPriority[MAX_GPUS] = { NULL, NULL };
float *cudaCurrentForwardProjection[MAX_GPUS] = { NULL, NULL };


__device__ float sampleScatterFactor(int mat, float x) {
  // cell centered to node centered indexing with normalized coordinates
  x = x*((MATERIAL_ANGULAR_SAMPLES-1.0f)/MATERIAL_ANGULAR_SAMPLES) + (0.5f/MATERIAL_ANGULAR_SAMPLES);

  switch (mat) {
  case 0:
    return tex1D(cudaMaterialTextures0, x);
  case 1:
    return tex1D(cudaMaterialTextures1, x);
  case 2:
    return tex1D(cudaMaterialTextures2, x);
  }
  return 0;
}


inline __host__ __device__ float3 operator-(float3 a, int3 b)
{
  return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}
inline __host__ __device__ float3 operator-(int3 a, float3 b)
{
  return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

inline __host__ __device__ float3 operator/(float3 a, int3 b)
{
  return make_float3(a.x / b.x, a.y / b.y, a.z / b.z);
}


__device__ float sampleSourceAttenuation(int combo, float3 x, int3 volDim) {
  // normalize coordinate
  x = x / (volDim-make_int3(1));

  // cell centered to node centered indexing with normalized coordinates
  x = x*((volDim-make_float3(1.0f))/volDim) + (make_float3(0.5f)/volDim);

  switch (combo) {
  case 0:  return tex3D(cudaSourceTextures00, x.x, x.y, x.z);
  case 1:  return tex3D(cudaSourceTextures01, x.x, x.y, x.z);
  case 2:  return tex3D(cudaSourceTextures02, x.x, x.y, x.z);
  case 3:  return tex3D(cudaSourceTextures10, x.x, x.y, x.z);
  case 4:  return tex3D(cudaSourceTextures11, x.x, x.y, x.z);
  case 5:  return tex3D(cudaSourceTextures12, x.x, x.y, x.z);
  case 6:  return tex3D(cudaSourceTextures20, x.x, x.y, x.z);
  case 7:  return tex3D(cudaSourceTextures21, x.x, x.y, x.z);
  case 8:  return tex3D(cudaSourceTextures22, x.x, x.y, x.z);
  }
  return 0; // should never get here
}


__device__ void sampleMaterialsAtPoint(int combo, float3 x, int3 volDim, int3 *mats, float3 *conc) {
  // normalize coordinate
  x = x / (volDim-make_int3(1));

  // cell centered to node centered indexing with normalized coordinates
  x = x*((volDim-make_float3(1.0f))/volDim) + (make_float3(0.5f)/volDim);

  float4 sample;
  switch (combo) {
  case 0:   sample = tex3D(cudaVolumeTextures00, x.x, x.y, x.z);  break;
  case 1:   sample = tex3D(cudaVolumeTextures01, x.x, x.y, x.z);  break;
  case 2:   sample = tex3D(cudaVolumeTextures02, x.x, x.y, x.z);  break;
  case 3:   sample = tex3D(cudaVolumeTextures10, x.x, x.y, x.z);  break;
  case 4:   sample = tex3D(cudaVolumeTextures11, x.x, x.y, x.z);  break;
  case 5:   sample = tex3D(cudaVolumeTextures12, x.x, x.y, x.z);  break;
  case 6:   sample = tex3D(cudaVolumeTextures20, x.x, x.y, x.z);  break;
  case 7:   sample = tex3D(cudaVolumeTextures21, x.x, x.y, x.z);  break;
  case 8:   sample = tex3D(cudaVolumeTextures22, x.x, x.y, x.z);  break;
  }


  // sort the concentrations so highest concentration is at .x, second is at .y
  (*mats) = make_int3(0,1,2);
  (*conc) = make_float3(sample.x, sample.y, sample.z);

  if (conc->x < conc->y) {
    float ti = mats->x;
    mats->x = mats->y;
    mats->y = ti;

    float tf = conc->x;
    conc->x = conc->y;
    conc->y = tf;
  }

  if (conc->y < conc->z) {
    float ti = mats->y;
    mats->y = mats->z;
    mats->z = ti;

    float tf = conc->y;
    conc->y = conc->z;
    conc->z = tf;
  }

  if (conc->x < conc->y) {
    float ti = mats->x;
    mats->x = mats->y;
    mats->y = ti;

    float tf = conc->x;
    conc->x = conc->y;
    conc->y = tf;
  }
}


//==================================================================================================
//==================================================================================================
//==================================================================================================
__global__ void castSourceRays(const float *cudaSourceScale,
                               const float3 *cudaSourceRayVolOrigin,
                               const float3 *cudaSourceRayVolDir,
                               const float *cudaSourceRayTMin,
                               const float *cudaSourceRayTMax,
                               int3 volDim,
                               int collectionStart,
                               int collectionEnd,
                               float3 matAtten,
                               float3 matDensity,
                               float voxelStepSize,
                               float *cudaSourceAttenuationCollection) {

  unsigned int gx = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned int gy = blockIdx.y*blockDim.y + threadIdx.y;
  unsigned int gz = blockIdx.z*blockDim.z + threadIdx.z;
  if (gx>=volDim.x || gy>=volDim.y || 
      gz<volDim.z*collectionStart || gz>=volDim.z*collectionEnd)
    return;
  unsigned int rayIndex = (gz%volDim.z)*volDim.x*volDim.y + gy*volDim.x + gx;
  int combo = gz/volDim.z;


  float3 volOrigin = cudaSourceRayVolOrigin[rayIndex];
  float3 volDir = cudaSourceRayVolDir[rayIndex];
  float tmin = cudaSourceRayTMin[rayIndex];
  float tmax = cudaSourceRayTMax[rayIndex];

  if (tmin>tmax) {
    // no attenuation from the volume
    cudaSourceAttenuationCollection[combo*volDim.x*volDim.y*volDim.z + rayIndex] = cudaSourceScale[rayIndex];
    return;
  }

  float3 volP0 = volOrigin + tmin*volDir;
  float3 volP1 = volOrigin + tmax*volDir;

  float volLength = length(volP1-volP0);
  int numSteps = volLength / voxelStepSize + 1;


  float lastInterfaceT = tmin;
  int3 lastMat;
  float3 lastConc;

  float3 matLens = make_float3(0,0,0);

  for (int step=0; step<numSteps; step++) {
    float samplef = (step+0.5f) / numSteps;
    float3 volSample = lerp(volP0, volP1, samplef);

    int3 thisMat;
    float3 thisConc;
    sampleMaterialsAtPoint(combo, volSample, volDim, &thisMat, &thisConc);

    // integrate between last step and this one
    if (step > 0) {
      float interfacef;

      // interpolate an interface
      if (thisMat.x != lastMat.x) {
        interfacef = (lastConc.x-lastConc.y) / ((thisConc.x-lastConc.y) - (thisConc.y-lastConc.x));
        interfacef = clamp(interfacef, 0.0f, 1.0f);
        interfacef = lerp((step-0.5f) / numSteps, samplef, interfacef);
      }

      // fixed interface
      else {
        interfacef = (float)step / numSteps;
      }

      float interfaceT = lerp(tmin, tmax, interfacef);

      // add material length before current interface
      float len = interfaceT - lastInterfaceT;
      switch (lastMat.x) {
      case 0:  matLens.x += len;  break;
      case 1:  matLens.y += len;  break;
      case 2:  matLens.z += len;  break;
      }

      lastInterfaceT = interfaceT;
    }

    lastMat = thisMat;
    lastConc = thisConc;
  }

  // add final material length
  float len = tmax - lastInterfaceT;
  switch (lastMat.x) {
  case 0:  matLens.x += len;  break;
  case 1:  matLens.y += len;  break;
  case 2:  matLens.z += len;  break;
  }


  // intergrate attenuations
  cudaSourceAttenuationCollection[combo*volDim.x*volDim.y*volDim.z + rayIndex] =
    cudaSourceScale[rayIndex] * 
    exp(-matLens.x * matDensity.x * matAtten.x +
        -matLens.y * matDensity.y * matAtten.y +
        -matLens.z * matDensity.z * matAtten.z);
}
                               


void MarkovContext::CudaComputeSourceAttenuation(int collectionSize) const {
  TIMER_START("CudaComputeSourceAttenuation()");

  int3 volDim = make_int3(mGeometry.GetVolumeNodeSamplesX(),
                          mGeometry.GetVolumeNodeSamplesY(),
                          mGeometry.GetVolumeNodeSamplesZ());

  float3 matAtten = make_float3(mMaterials[0].GetMassAttenuationCoefficient(),
                                mMaterials[1].GetMassAttenuationCoefficient(),
                                mMaterials[2].GetMassAttenuationCoefficient());
  float3 matDensity = make_float3(mMaterials[0].GetDensity(),
                                  mMaterials[1].GetDensity(),
                                  mMaterials[2].GetDensity());
                                

  dim3 dimBlock(8, 8, 1);
  dim3 dimGrid(1+(volDim.x-1) / dimBlock.x, 
               1+(volDim.y-1) / dimBlock.y, 
               1+(collectionSize*volDim.z-1) / dimBlock.z);

  for (int g=0; g<mGpuIds.size(); g++) {
    int collectionStart, collectionEnd;
    CudaGetCollectionStartEnd(g, collectionSize, collectionStart, collectionEnd);

    checkCudaErrors(hipSetDevice(mGpuIds[g]));
    castSourceRays<<<dimGrid, dimBlock>>>(cudaSourceScale[g],
                                          cudaSourceRayVolOrigin[g],
                                          cudaSourceRayVolDir[g],
                                          cudaSourceRayTMin[g],
                                          cudaSourceRayTMax[g],
                                          volDim,
                                          collectionStart,
                                          collectionEnd,
                                          matAtten,
                                          matDensity,
                                          mVoxelStepSize,
                                          cudaSourceAttenuationCollection[g]);

    for (int c=collectionStart; c<collectionEnd; c++) {
      hipArray **hipArray = NULL;
      switch (c) {
      case 0:
        hipArray = &cudaSourceArray00[g];
        break;
      case 1:
        hipArray = &cudaSourceArray01[g];
        break;
      case 2:
        hipArray = &cudaSourceArray02[g];
        break;
      case 3:
        hipArray = &cudaSourceArray10[g];
        break;
      case 4:
        hipArray = &cudaSourceArray11[g];
        break;
      case 5:
        hipArray = &cudaSourceArray12[g];
        break;
      case 6:
        hipArray = &cudaSourceArray20[g];
        break;
      case 7:
        hipArray = &cudaSourceArray21[g];
        break;
      case 8:
        hipArray = &cudaSourceArray22[g];
        break;
      }


      hipExtent volumeSize = make_hipExtent(mGeometry.GetVolumeNodeSamplesX(),
                                              mGeometry.GetVolumeNodeSamplesY(),
                                              mGeometry.GetVolumeNodeSamplesZ());
      // copy data to 3D array
      hipMemcpy3DParms copyParams = {0};
      copyParams.srcPtr   = make_hipPitchedPtr((void *)&cudaSourceAttenuationCollection[g][mGeometry.GetTotalVolumeNodeSamples() * c],
                                                volumeSize.width*sizeof(float), volumeSize.width, volumeSize.height);
      copyParams.dstArray = *hipArray;
      copyParams.extent   = volumeSize;
      copyParams.kind     = hipMemcpyDeviceToDevice;
      checkCudaErrors(hipMemcpy3DAsync(&copyParams));
    }
  }

  TIMER_STOP;
}

void MarkovContext::CudaGetSourceAttenuation(int collectionSize,
                                             vector< vector<float> > &sourceAttenuationCollection) const {
  // copy results back 
  vector<float> outputData(collectionSize * mGeometry.GetTotalVolumeNodeSamples());
  for (int g=0; g<mGpuIds.size(); g++) {
    int collectionStart, collectionEnd;
    CudaGetCollectionStartEnd(g, collectionSize, collectionStart, collectionEnd);

    checkCudaErrors(hipSetDevice(mGpuIds[g]));
    if ((collectionEnd-collectionStart) > 0) {
      checkCudaErrors(hipMemcpy(&outputData[collectionStart * mGeometry.GetTotalVolumeNodeSamples()],
                                 &cudaSourceAttenuationCollection[g][collectionStart * mGeometry.GetTotalVolumeNodeSamples()],
                                 (collectionEnd-collectionStart) * mGeometry.GetTotalVolumeNodeSamples()*sizeof(float),
                                 hipMemcpyDeviceToHost));
    }
  }

  sourceAttenuationCollection.resize(collectionSize);
  for (int c=0; c<collectionSize; c++) {
    sourceAttenuationCollection[c].resize(mCurrentVolumeSourceAttenuation.size());
    for (int i=0; i<mCurrentVolumeSourceAttenuation.size(); i++)
      sourceAttenuationCollection[c][i] = outputData[c*mGeometry.GetTotalVolumeNodeSamples() + i];
  }
}




//==================================================================================================
//==================================================================================================
//==================================================================================================

__global__ void prioritizeDetectorRays(const float3 *cudaDetectorRayWorldOrigin,
                                       const float3 *cudaDetectorRayWorldDir,
                                       const float *cudaDetectorRayTMin,
                                       const float *cudaDetectorRayTMax,
                                       int numRays,
                                       float3 coneOrigin,
                                       float3 coneDir,
                                       float coneCosTheta,
                                       float coneMinDist,
                                       unsigned int *cudaRayIds,
                                       float *cudaRayPriority) {
  unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx >= numRays)
    return;

  float3 rayOrigin = cudaDetectorRayWorldOrigin[idx];
  float3 rayDir = cudaDetectorRayWorldDir[idx];


  float AdD = dot(coneDir, rayDir);
  float cosSqr = (coneCosTheta-COSTHETA_EPS)*(coneCosTheta-COSTHETA_EPS);
  float3 E = rayOrigin - coneOrigin;
  float AdE = dot(coneDir, E);
  float DdE = dot(rayDir, E);
  float EdE = dot(E, E);
  float c2 = AdD*AdD - cosSqr;
  float c1 = AdD*AdE - cosSqr*DdE;
  float c0 = AdE*AdE - cosSqr*EdE;
  float dp;

  float3 point;
  bool hit = false;

  // Solve the quadratic.  Keep only those X for which Dot(A,X-V) >= 0.
  if (fabsf(c2) >= 1e-4) {
    // c2 != 0
    float discr = c1*c1 - c0*c2;
    if (discr > 1e-4) {
      // Q(t) = 0 has two distinct real-valued roots.  However, one or
      // both of them might intersect the portion of the double-sided
      // cone "behind" the vertex.  We are interested only in those
      // intersections "in front" of the vertex.
      float root = sqrtf(discr);
      float invC2 = 1.0f/c2;

      float t = (-c1 - root)*invC2;
      point = rayOrigin + t*rayDir;
      E = point - coneOrigin;
      dp = dot(E, coneDir);
      if (dp > coneMinDist-MINDIST_EPS) {
        hit = true;
      }

      t = (-c1 + root)*invC2;
      point = rayOrigin + t*rayDir;
      E = point - coneOrigin;
      dp = dot(E, coneDir);
      if (dp > coneMinDist-MINDIST_EPS) {
        hit = true;
      }
    }
  }

  //  hit = true;

  if (hit)
    cudaRayPriority[idx] = 1;//(cudaDetectorRayTMax[idx] - cudaDetectorRayTMin[idx]);
  else
    cudaRayPriority[idx] = 0;

  cudaRayIds[idx] = idx;
}


__device__ void castDetectorRay(int rayIndex, int combo,
                                const float3 *cudaDetectorRayWorldOrigin,
                                const float3 *cudaDetectorRayWorldDir,
                                const float3 *cudaDetectorRayVolOrigin,
                                const float3 *cudaDetectorRayVolDir,
                                const float *cudaDetectorRayTMin,
                                const float *cudaDetectorRayTMax,
                                const float *cudaProjectionAngles,
                                int3 detectorDim,
                                int3 volDim,
                                float3 matAtten,
                                float3 matDensity,
                                float3 sourcePosition,
                                float voxelStepSize,
                                float *cudaForwardProjectionCollection) {

  float3 worldOrigin = cudaDetectorRayWorldOrigin[rayIndex];
  float3 worldDir = cudaDetectorRayWorldDir[rayIndex];
  float3 volOrigin = cudaDetectorRayVolOrigin[rayIndex];
  float3 volDir = cudaDetectorRayVolDir[rayIndex];
  float tmin = cudaDetectorRayTMin[rayIndex];
  float tmax = cudaDetectorRayTMax[rayIndex];

  if (tmin>tmax) {
    // no attenuation from the volume
    cudaForwardProjectionCollection[combo*detectorDim.x*detectorDim.y*detectorDim.z + rayIndex] = 0;
    return;
  }

  float3 volP0 = volOrigin + tmin*volDir;
  float3 volP1 = volOrigin + tmax*volDir;

  // compute ray trajectory in rotated worldspace coordinates for source falloff lookup
  // assumes detector spacing of 10 degrees!
  float theta = cudaProjectionAngles[rayIndex/(detectorDim.x*detectorDim.y)];
  float ctheta = cosf(-theta); // backwards rotation
  float stheta = sinf(-theta);
  float3 worldSourceP0 = worldOrigin + tmin*worldDir;
  float3 worldSourceP1 = worldOrigin + tmax*worldDir;
  worldSourceP0 = make_float3(ctheta*worldSourceP0.x - stheta*worldSourceP0.y,
                               stheta*worldSourceP0.x + ctheta*worldSourceP0.y,
                               worldSourceP0.z);
  worldSourceP1 = make_float3(ctheta*worldSourceP1.x - stheta*worldSourceP1.y,
                               stheta*worldSourceP1.x + ctheta*worldSourceP1.y,
                               worldSourceP1.z);
  worldSourceP0.x = -worldSourceP0.x;
  worldSourceP1.x = -worldSourceP1.x;
  worldSourceP0.z = sourcePosition.z - worldSourceP0.z;
  worldSourceP1.z = sourcePosition.z - worldSourceP1.z;

  float volLength = length(volP1-volP0);
  int numSteps = volLength / voxelStepSize + 1;


  int3 lastMat;
  float3 lastConc;
  float lastInterfaceT = tmin;

  float forwardProjection = 0;
  float sumDetectorAttenuation = 0;

  float lastInterfaceSourceAtten = sampleSourceAttenuation(combo, volP0, volDim);
  lastInterfaceSourceAtten *= tex2D(cudaSourceFalloffTexture, 
                                    worldSourceP0.x/worldSourceP0.z * (0.19/0.205) + 0.5,
                                    worldSourceP0.y/worldSourceP0.z * (0.19/0.205) + 0.5);


  for (int step=0; step<numSteps; step++) {
    float samplef = (step+0.5f) / numSteps;
    float3 volSample = lerp(volP0, volP1, samplef);

    int3 thisMat;
    float3 thisConc;
    sampleMaterialsAtPoint(combo, volSample, volDim, &thisMat, &thisConc);

    // integrate between last step and this one
    if (step > 0) {
      float interfacef;

      // interpolate an interface
      if (thisMat.x != lastMat.x) {
        interfacef = (lastConc.x-lastConc.y) / ((thisConc.x-lastConc.y) - (thisConc.y-lastConc.x));
        interfacef = clamp(interfacef, 0.0f, 1.0f);
        interfacef = lerp((step-0.5f) / numSteps, samplef, interfacef);
      }

      // fixed interface
      else {
        interfacef = (float)step / numSteps;
      }

      float interfaceT = lerp(tmin, tmax, interfacef);

      //
      // integrate this step
      //
      float tdist = interfaceT - lastInterfaceT;

      // direction to center of material segment from source
      float3 sourceRayDir = normalize((worldOrigin + ((interfaceT+lastInterfaceT)*0.5f)*worldDir) - sourcePosition);
      float ncosScatterAngle = dot(worldDir, sourceRayDir);
      float thisInterfaceSourceAtten = sampleSourceAttenuation(combo, volOrigin + interfaceT*volDir, volDim);
      
      float3 worldSourceP = lerp(worldSourceP0, worldSourceP1, interfacef);
      thisInterfaceSourceAtten *= tex2D(cudaSourceFalloffTexture,
                                        worldSourceP.x/worldSourceP.z * (0.19/0.205) + 0.5,
                                        worldSourceP.y/worldSourceP.z * (0.19/0.205) + 0.5);
      

      float density = 0;
      float massAtten = 0;
      switch (lastMat.x) {
      case 0:
        density = matDensity.x;
        massAtten = matAtten.x;
        break;
      case 1:
        density = matDensity.y;
        massAtten = matAtten.y;
        break;
      case 2:
        density = matDensity.z;
        massAtten = matAtten.z;
        break;
      }

      float voxelAttenuation = -tdist * density * massAtten;

      float attenuationFactor;
      if (voxelAttenuation == 0) {
        attenuationFactor = (lastInterfaceSourceAtten+thisInterfaceSourceAtten)*0.5f;
      }
      else {
        double a = lastInterfaceSourceAtten;
        double b = thisInterfaceSourceAtten;
        double c = voxelAttenuation;
        attenuationFactor = (exp(c) * (a+b*(c-1)) - a*(c+1)+b) / (c*c);
      }

      // attenuation between detector and start
      attenuationFactor *= exp(sumDetectorAttenuation);

      forwardProjection += (tdist *
                            sampleScatterFactor(lastMat.x, 0.5f*(1+ncosScatterAngle)) *
                            attenuationFactor);


      sumDetectorAttenuation += voxelAttenuation;

      // store info for next step
      lastInterfaceT = interfaceT;
      lastInterfaceSourceAtten = thisInterfaceSourceAtten;
    }

    lastMat = thisMat;
    lastConc = thisConc;
  }


  //
  // integrate final step
  //
  float interfaceT = tmax;
  float tdist = interfaceT - lastInterfaceT;

  // direction to center of material segment from source
  float3 sourceRayDir = normalize((worldOrigin + ((interfaceT+lastInterfaceT)*0.5f)*worldDir) - sourcePosition);
  float ncosScatterAngle = dot(worldDir, sourceRayDir);
  float thisInterfaceSourceAtten = sampleSourceAttenuation(combo, volOrigin + interfaceT*volDir, volDim);
  thisInterfaceSourceAtten *= tex2D(cudaSourceFalloffTexture,
                                    worldSourceP1.x/worldSourceP1.z * (0.19/0.205) + 0.5,
                                    worldSourceP1.y/worldSourceP1.z * (0.19/0.205) + 0.5);

  float density = 0;
  float massAtten = 0;
  switch (lastMat.x) {
  case 0:
    density = matDensity.x;
    massAtten = matAtten.x;
    break;
  case 1:
    density = matDensity.y;
    massAtten = matAtten.y;
    break;
  case 2:
    density = matDensity.z;
    massAtten = matAtten.z;
    break;
  }

  float voxelAttenuation = -tdist * density * massAtten;

  float attenuationFactor;
  if (voxelAttenuation == 0) {
    attenuationFactor = (lastInterfaceSourceAtten+thisInterfaceSourceAtten)*0.5f;
  }
  else {
    double a = lastInterfaceSourceAtten;
    double b = thisInterfaceSourceAtten;
    double c = voxelAttenuation;
    attenuationFactor = (exp(c) * (a+b*(c-1)) - a*(c+1)+b) / (c*c);
  }

  // attenuation between detector and start
  attenuationFactor *= exp(sumDetectorAttenuation);

  forwardProjection += (tdist *
                        (0.5 * (1+sampleScatterFactor(lastMat.x, ncosScatterAngle))) *
                        attenuationFactor);



  // store result
  cudaForwardProjectionCollection[combo*detectorDim.x*detectorDim.y*detectorDim.z + rayIndex] = forwardProjection;
}
                               

__global__ void castAllDetectorRays(const float3 *cudaDetectorRayWorldOrigin,
                                    const float3 *cudaDetectorRayWorldDir,
                                    const float3 *cudaDetectorRayVolOrigin,
                                    const float3 *cudaDetectorRayVolDir,
                                    const float *cudaDetectorRayTMin,
                                    const float *cudaDetectorRayTMax,
                                    const float *cudaProjectionAngles,
                                    int3 detectorDim,
                                    int3 volDim,
                                    int collectionStart,
                                    int collectionEnd,
                                    float3 matAtten,
                                    float3 matDensity,
                                    float3 sourcePosition,
                                    float voxelStepSize,
                                    float *cudaForwardProjectionCollection) {

  unsigned int gx = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned int gy = blockIdx.y*blockDim.y + threadIdx.y;
  unsigned int gz = blockIdx.z*blockDim.z + threadIdx.z;
  if (gx>=detectorDim.x || gy>=detectorDim.y || 
      gz<detectorDim.z*collectionStart || gz>=detectorDim.z*collectionEnd)
    return;
  unsigned int rayIndex = (gz%detectorDim.z)*detectorDim.x*detectorDim.y + gy*detectorDim.x + gx;
  int combo = gz/detectorDim.z;

  castDetectorRay(rayIndex, combo,
                  cudaDetectorRayWorldOrigin,
                  cudaDetectorRayWorldDir,
                  cudaDetectorRayVolOrigin,
                  cudaDetectorRayVolDir,
                  cudaDetectorRayTMin,
                  cudaDetectorRayTMax,
                  cudaProjectionAngles,
                  detectorDim,
                  volDim,
                  matAtten,
                  matDensity,
                  sourcePosition,
                  voxelStepSize,
                  cudaForwardProjectionCollection);
}



__global__ void castPrioritizedDetectorRays(const unsigned int *cudaRayIds,
                                            const float *cudaRayPriority,
                                            const float *cudaCurrentForwardProjection,
                                            const float3 *cudaDetectorRayWorldOrigin,
                                            const float3 *cudaDetectorRayWorldDir,
                                            const float3 *cudaDetectorRayVolOrigin,
                                            const float3 *cudaDetectorRayVolDir,
                                            const float *cudaDetectorRayTMin,
                                            const float *cudaDetectorRayTMax,
                                            const float *cudaProjectionAngles,
                                            int3 detectorDim,
                                            int3 volDim,
                                            int collectionStart,
                                            int collectionEnd,
                                            float3 matAtten,
                                            float3 matDensity,
                                            float3 sourcePosition,
                                            float voxelStepSize,
                                            float *cudaForwardProjectionCollection) {

  unsigned int gx = blockIdx.x*blockDim.x + threadIdx.x;
  if (gx<detectorDim.x*detectorDim.y*detectorDim.z*collectionStart ||
      gx>=detectorDim.x*detectorDim.y*detectorDim.z*collectionEnd)
    return;

  //unsigned int rayIndexIndex = gx % (detectorDim.x*detectorDim.y*detectorDim.z);
  //int combo = gx / (detectorDim.x*detectorDim.y*detectorDim.z);
  unsigned int rayIndexIndex = gx % (detectorDim.x*detectorDim.y*detectorDim.z);
  int combo = gx / (detectorDim.x*detectorDim.y*detectorDim.z);

  unsigned int rayIndex = cudaRayIds[rayIndexIndex];
  float priority = cudaRayPriority[rayIndexIndex];

  if (priority == 0) {
    cudaForwardProjectionCollection[combo*
                                    detectorDim.x*
                                    detectorDim.y*
                                    detectorDim.z +
                                    rayIndex] = cudaCurrentForwardProjection[rayIndex];
  }
  else {
    castDetectorRay(rayIndex, combo,
                    cudaDetectorRayWorldOrigin,
                    cudaDetectorRayWorldDir,
                    cudaDetectorRayVolOrigin,
                    cudaDetectorRayVolDir,
                    cudaDetectorRayTMin,
                    cudaDetectorRayTMax,
                    cudaProjectionAngles,
                    detectorDim,
                    volDim,
                    matAtten,
                    matDensity,
                    sourcePosition,
                    voxelStepSize,
                    cudaForwardProjectionCollection);
  }
}




void MarkovContext::CudaForwardProject(int collectionSize) const {
  TIMER_START("CudaForwardProject()");

  int3 volDim = make_int3(mGeometry.GetVolumeNodeSamplesX(),
                          mGeometry.GetVolumeNodeSamplesY(),
                          mGeometry.GetVolumeNodeSamplesZ());
  int3 detectorDim = make_int3(mGeometry.GetDetectorSamplesWidth(),
                               mGeometry.GetDetectorSamplesHeight(),
                               mGeometry.GetNumProjectionAngles());

  float3 matAtten = make_float3(mMaterials[0].GetMassAttenuationCoefficient(),
                                mMaterials[1].GetMassAttenuationCoefficient(),
                                mMaterials[2].GetMassAttenuationCoefficient());
  float3 matDensity = make_float3(mMaterials[0].GetDensity(),
                                  mMaterials[1].GetDensity(),
                                  mMaterials[2].GetDensity());

  float3 sourcePosition = make_float3(mGeometry.GetSourcePosition()[0],
                                      mGeometry.GetSourcePosition()[1],
                                      mGeometry.GetSourcePosition()[2]);
                                

  dim3 dimBlock(8, 8, 1);
  dim3 dimGrid(1+(detectorDim.x-1) / dimBlock.x, 
               1+(detectorDim.y-1) / dimBlock.y, 
               1+(collectionSize*detectorDim.z-1) / dimBlock.z);

  for (int g=0; g<mGpuIds.size(); g++) {
    int collectionStart, collectionEnd;
    CudaGetCollectionStartEnd(g, collectionSize, collectionStart, collectionEnd);

    checkCudaErrors(hipSetDevice(mGpuIds[g]));
    castAllDetectorRays<<<dimGrid, dimBlock>>>(cudaDetectorRayWorldOrigin[g],
                                               cudaDetectorRayWorldDir[g],
                                               cudaDetectorRayVolOrigin[g],
                                               cudaDetectorRayVolDir[g],
                                               cudaDetectorRayTMin[g],
                                               cudaDetectorRayTMax[g],
                                               cudaProjectionAngles[g],
                                               detectorDim,
                                               volDim,
                                               collectionStart,
                                               collectionEnd,
                                               matAtten,
                                               matDensity,
                                               sourcePosition,
                                               mVoxelStepSize,
                                               cudaForwardProjectionCollection[g]);
  }

  TIMER_STOP;
}


#ifdef CUDA_ENABLE_UPDATE_FORWARD_PROJECTION
void MarkovContext::CudaUpdateForwardProjection(int collectionSize,
                                                const Cone &attenChangeCone) const {

  if (mGpuIds.size() > 1) {
    // currently, forward projections are not copied between gpus when a config is accepted
    std::cerr<<"CudaUpdateForwardProjection() currently does not support more than one GPU!"<<std::endl;
    return;
  }

  TIMER_START("CudaUpdateForwardProjection()");

  int3 volDim = make_int3(mGeometry.GetVolumeNodeSamplesX(),
                          mGeometry.GetVolumeNodeSamplesY(),
                          mGeometry.GetVolumeNodeSamplesZ());
  int3 detectorDim = make_int3(mGeometry.GetDetectorSamplesWidth(),
                               mGeometry.GetDetectorSamplesHeight(),
                               mGeometry.GetNumProjectionAngles());

  float3 matAtten = make_float3(mMaterials[0].GetMassAttenuationCoefficient(),
                                mMaterials[1].GetMassAttenuationCoefficient(),
                                mMaterials[2].GetMassAttenuationCoefficient());
  float3 matDensity = make_float3(mMaterials[0].GetDensity(),
                                  mMaterials[1].GetDensity(),
                                  mMaterials[2].GetDensity());

  float3 sourcePosition = make_float3(mGeometry.GetSourcePosition()[0],
                                      mGeometry.GetSourcePosition()[1],
                                      mGeometry.GetSourcePosition()[2]);

  // prioritize each ray
  dim3 dimBlockPrioritize(32,1,1);
  dim3 dimGridPrioritize(1+(mGeometry.GetTotalProjectionSamples()-1) / dimBlockPrioritize.x, 1, 1);
  for (int g=0; g<mGpuIds.size(); g++) {
    checkCudaErrors(hipSetDevice(mGpuIds[g]));
    prioritizeDetectorRays<<<dimGridPrioritize, dimBlockPrioritize>>>(cudaDetectorRayWorldOrigin[g],
                                                                      cudaDetectorRayWorldDir[g],
                                                                      cudaDetectorRayTMin[g],
                                                                      cudaDetectorRayTMax[g],
                                                                      mGeometry.GetTotalProjectionSamples(),
                                                                      make_float3(attenChangeCone.mOrigin[0],
                                                                                  attenChangeCone.mOrigin[1],
                                                                                  attenChangeCone.mOrigin[2]),
                                                                      make_float3(attenChangeCone.mDir[0],
                                                                                  attenChangeCone.mDir[1],
                                                                                  attenChangeCone.mDir[2]),
                                                                      attenChangeCone.mCosTheta,
                                                                      attenChangeCone.mMinDist,
                                                                      cudaRayIds[g],
                                                                      cudaRayPriority[g]);
  }

  // sort rays by priority
  for (int g=0; g<mGpuIds.size(); g++) {
    checkCudaErrors(hipSetDevice(mGpuIds[g]));
    thrust::device_ptr<unsigned int> thrustIds = thrust::device_pointer_cast(cudaRayIds[g]);
    thrust::device_ptr<float> thrustPriorities = thrust::device_pointer_cast(cudaRayPriority[g]);
    thrust::sort_by_key(thrustPriorities, thrustPriorities+mGeometry.GetTotalProjectionSamples(), thrustIds);
    //thrust::stable_sort_by_key(thrustPriorities, thrustPriorities+mGeometry.GetTotalProjectionSamples(), thrustIds);

    /*
      vector<float> priorities(mGeometry.GetTotalProjectionSamples());
      checkCudaErrors(hipMemcpy(&priorities[0], cudaRayPriority, sizeof(float)*mGeometry.GetTotalProjectionSamples(), hipMemcpyDeviceToHost));

      int skippedRays = 0;
      for (int i=0; i<priorities.size(); i++) {
      if (priorities[i] == 0) {
      skippedRays++;
      }
      }

      std::cerr<<"skipped "<<skippedRays<<" of "<<priorities.size()<<std::endl;
    */


    // cast only non-zero priority rays
    /*
      dim3 dimBlock(8, 8, 1);
      dim3 dimGrid(1+(detectorDim.x-1) / dimBlock.x, 
      1+(detectorDim.y-1) / dimBlock.y, 
      1+(collectionSize*detectorDim.z-1) / dimBlock.z);
    */
  }


  dim3 dimBlock(32,1,1);
  dim3 dimGrid(1+(collectionSize * mGeometry.GetTotalProjectionSamples()-1) / dimBlockPrioritize.x, 1, 1);

  for (int g=0; g<mGpuIds.size(); g++) {
    int collectionStart, collectionEnd;
    CudaGetCollectionStartEnd(g, collectionSize, collectionStart, collectionEnd);

    checkCudaErrors(hipSetDevice(mGpuIds[g]));
    castPrioritizedDetectorRays<<<dimGrid, dimBlock>>>(cudaRayIds[g],
                                                       cudaRayPriority[g],
                                                       cudaCurrentForwardProjection[g],
                                                       cudaDetectorRayWorldOrigin[g],
                                                       cudaDetectorRayWorldDir[g],
                                                       cudaDetectorRayVolOrigin[g],
                                                       cudaDetectorRayVolDir[g],
                                                       cudaDetectorRayTMin[g],
                                                       cudaDetectorRayTMax[g],
                                                       cudaProjectionAngles[g],
                                                       detectorDim,
                                                       volDim,
                                                       collectionStart,
                                                       collectionEnd,
                                                       matAtten,
                                                       matDensity,
                                                       sourcePosition,
                                                       mVoxelStepSize,
                                                       cudaForwardProjectionCollection[g]);
  }

  TIMER_STOP;
}
#endif

void MarkovContext::CudaGetForwardProjection(int collectionSize,
                                             vector< vector<float> > &forwardProjectionCollection) const {
  TIMER_START("CudaGetForwardProjection()");

  // copy results back
  //vector<float> outputData(collectionSize * mGeometry.GetTotalProjectionSamples());
  float *outputData;
  checkCudaErrors(hipHostAlloc(&outputData,
                                collectionSize * mGeometry.GetTotalProjectionSamples() * sizeof(float),
                                hipHostMallocDefault));

  for (int g=0; g<mGpuIds.size(); g++) {
    int collectionStart, collectionEnd;
    CudaGetCollectionStartEnd(g, collectionSize, collectionStart, collectionEnd);

    checkCudaErrors(hipSetDevice(mGpuIds[g]));
    if ((collectionEnd-collectionStart) > 0) {
      checkCudaErrors(hipMemcpyAsync(&outputData[collectionStart * mGeometry.GetTotalProjectionSamples()],
                                      &cudaForwardProjectionCollection[g][collectionStart * mGeometry.GetTotalProjectionSamples()],
                                      (collectionEnd-collectionStart) * mGeometry.GetTotalProjectionSamples()*sizeof(float),
                                      hipMemcpyDeviceToHost));
    }
  }

  for (int g=0; g<mGpuIds.size(); g++) {
    checkCudaErrors(hipSetDevice(mGpuIds[g]));
    checkCudaErrors(hipDeviceSynchronize());
  }


  forwardProjectionCollection.resize(collectionSize);
  for (int c=0; c<collectionSize; c++) {
    forwardProjectionCollection[c].resize(mGeometry.GetTotalProjectionSamples());
    for (int i=0; i<mGeometry.GetTotalProjectionSamples(); i++) {
      forwardProjectionCollection[c][i] = outputData[c*mGeometry.GetTotalProjectionSamples() + i];
    }
  }

  checkCudaErrors(hipHostFree(outputData));


  TIMER_STOP;
}



//==================================================================================================
//==================================================================================================
//==================================================================================================
__global__ void projectionToError(int totalProjectionSamples,
                                  int collectionSize,
                                  const float *baselineProjection,
                                  const float *forwardProjection,
                                  float *forwardProjectionError) {

  unsigned int cpi = blockIdx.x*blockDim.x + threadIdx.x;
  if (cpi >= totalProjectionSamples*collectionSize)
    return;

  int pi = cpi % totalProjectionSamples;
  float df = forwardProjection[cpi] - baselineProjection[pi];
  forwardProjectionError[cpi] = df*df;
}


void MarkovContext::CudaGetProjectionError(int collectionSize, vector<float> &errors) const {

  TIMER_START("CudaGetProjectionError()");

  // compute squared errors
  int totalProjectionSamples = mGeometry.GetTotalProjectionSamples();
  dim3 dimBlock(32, 1, 1);
  dim3 dimGrid(1+((totalProjectionSamples*collectionSize)-1) / dimBlock.x, 1, 1);

  for (int g=0; g<mGpuIds.size(); g++) {
    checkCudaErrors(hipSetDevice(mGpuIds[g]));
    projectionToError<<<dimGrid, dimBlock>>>(totalProjectionSamples,
                                             collectionSize,
                                             cudaBaselineProjection[g],
                                             cudaForwardProjectionCollection[g],
                                             cudaForwardProjectionError[g]);
  }

  // use thrust to sum the errors for each material combo
  for (int g=0; g<mGpuIds.size(); g++) {
    checkCudaErrors(hipSetDevice(mGpuIds[g]));
    thrust::device_ptr<float> dev_ptr = thrust::device_pointer_cast(cudaForwardProjectionError[g]);
    errors.resize(collectionSize);
    for (int c=0; c<collectionSize; c++) {
      errors[c] = (thrust::reduce(dev_ptr+c*totalProjectionSamples,
                                  dev_ptr+(c+1)*totalProjectionSamples)
                   * mGeometry.GetDetectorPixelArea());
    }
  }

  TIMER_STOP;
}



//==================================================================================================
//==================================================================================================
//==================================================================================================
__global__ void cudaUpdateVolume1(float4 *vol,
                                  int idx,
                                  int mat) {

  // only a single thread needs to do anything
  if (blockIdx.x != 0 || threadIdx.x != 0)
    return;

  float4 nv = make_float4(0,0,0,0);
  switch (mat) {
  case 0:  nv.x = 1;  break;
  case 1:  nv.y = 1;  break;
  case 2:  nv.z = 1;  break;
  }

  vol[idx] = nv;
}


__global__ void cudaUpdateVolume2(float4 *vol,
                                  int idx,
                                  int mat,
                                  int idx2,
                                  int mat2) {

  // only a single thread needs to do anything
  if (blockIdx.x != 0 || threadIdx.x != 0)
    return;

  float4 nv = make_float4(0,0,0,0);
  switch (mat) {
  case 0:  nv.x = 1;  break;
  case 1:  nv.y = 1;  break;
  case 2:  nv.z = 1;  break;
  }
  vol[idx] = nv;

  float4 nv2 = make_float4(0,0,0,0);
  switch (mat2) {
  case 0:  nv2.x = 1;  break;
  case 1:  nv2.y = 1;  break;
  case 2:  nv2.z = 1;  break;
  }
  vol[idx2] = nv2;
}



void MarkovContext::CudaAcceptNextConfig(const GibbsProposal &proposal, int c) const {
  TIMER_START("CudaAcceptNextConfig()");

  for (int g=0; g<mGpuIds.size(); g++) {
    int collectionStart, collectionEnd;
    CudaGetCollectionStartEnd(g, NUM_MATERIALS*NUM_MATERIALS, collectionStart, collectionEnd);

    checkCudaErrors(hipSetDevice(mGpuIds[g]));

    // update all collection volumes
    for (int c2=collectionStart; c2<collectionEnd; c2++) {
      float4 *cudaVolume = cudaVolumeLinearCollection[g] + c2*mGeometry.GetTotalVolumeNodeSamples();

      // apply changes
      // first proposal only
      if (proposal.first>=0 && proposal.second<0) {
        dim3 dimBlock(1,1,1);
        dim3 dimGrid(1,1,1);
        cudaUpdateVolume1<<<dimGrid, dimBlock>>>(cudaVolume,
                                                 proposal.first,
                                                 c%NUM_MATERIALS);
      }

      // apply changes
      // both proposals
      else if (proposal.first>=0 && proposal.second>=0) {
        dim3 dimBlock(1,1,1);
        dim3 dimGrid(1,1,1);
        cudaUpdateVolume2<<<dimGrid, dimBlock>>>(cudaVolume,
                                                 proposal.first,
                                                 c%NUM_MATERIALS,
                                                 proposal.second,
                                                 c/NUM_MATERIALS);
      }
    }


    // set the accepted forward projection as current
#ifdef CUDA_ENABLE_UPDATE_FORWARD_PROJECTION
    checkCudaErrors(hipMemcpyAsync(cudaCurrentForwardProjection[g],
                                    cudaForwardProjectionCollection[g] + c*mGeometry.GetTotalProjectionSamples(),
                                    sizeof(float)*mGeometry.GetTotalProjectionSamples(),
                                    hipMemcpyDeviceToDevice));
#endif
  }

  TIMER_STOP;
}


void MarkovContext::CudaSetCurrentVolume(const vector<unsigned char> &matids) const {

  // setup individual channels per material
  vector<float4> volumeData(mGeometry.GetTotalVolumeNodeSamples());
  memset(&volumeData[0], 0, sizeof(float4)*mGeometry.GetTotalVolumeNodeSamples());
  for (int i=0; i<mGeometry.GetTotalVolumeNodeSamples(); i++) {
    switch (matids[i]) {
    case 0:  volumeData[i].x = 1;  break;
    case 1:  volumeData[i].y = 1;  break;
    case 2:  volumeData[i].z = 1;  break;
    }
  }

  for (int g=0; g<mGpuIds.size(); g++) {
    checkCudaErrors(hipSetDevice(mGpuIds[g]));

    // set all collection volumes
    for (int c=0; c<NUM_MATERIALS*NUM_MATERIALS; c++) {
      checkCudaErrors(hipMemcpy(cudaVolumeLinearCollection[g] + c*mGeometry.GetTotalVolumeNodeSamples(),
                                 &volumeData[0],
                                 sizeof(float4)*mGeometry.GetTotalVolumeNodeSamples(),
                                 hipMemcpyHostToDevice));
    }
  }
}


void MarkovContext::CudaGetVolumeCollection(vector< vector<unsigned char> > &volumeCollection) const {
  volumeCollection.resize(NUM_MATERIALS*NUM_MATERIALS);
  for (int c=0; c<NUM_MATERIALS*NUM_MATERIALS; c++) {

    vector<float4> fvol(mGeometry.GetTotalVolumeNodeSamples());

    checkCudaErrors(hipMemcpy(&fvol[0],
                               cudaVolumeLinearCollection + c*mGeometry.GetTotalVolumeNodeSamples(),
                               sizeof(float4)*mGeometry.GetTotalVolumeNodeSamples(),
                               hipMemcpyDeviceToHost));


    volumeCollection[c].resize(mGeometry.GetTotalVolumeNodeSamples());
    for (int i=0; i<mGeometry.GetTotalVolumeNodeSamples(); i++) {
      if (fvol[i].x == 1)
        volumeCollection[c][i] = 0;
      else if (fvol[i].y == 1)
        volumeCollection[c][i] = 1;
      else if (fvol[i].z == 1)
        volumeCollection[c][i] = 2;
      else {
        std::cerr<<"bogus volume!"<<std::endl;
        exit(0);
      }
    }
    
  }
}


void MarkovContext::CudaSetVolumeCollection(const GibbsProposal &proposal) const {
  TIMER_START("CudaSetVolumeCollection()");

  for (int g=0; g<mGpuIds.size(); g++) {
    int collectionStart, collectionEnd;
    CudaGetCollectionStartEnd(g, NUM_MATERIALS*NUM_MATERIALS, collectionStart, collectionEnd);

    checkCudaErrors(hipSetDevice(mGpuIds[g]));
 
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
    hipExtent volumeSize = make_hipExtent(mGeometry.GetVolumeNodeSamplesX(),
                                            mGeometry.GetVolumeNodeSamplesY(),
                                            mGeometry.GetVolumeNodeSamplesZ());

    // apply changes to each collection volume
    for (int c=collectionStart; c<collectionEnd; c++) {
      float4 *cudaVolume = cudaVolumeLinearCollection[g] + c*mGeometry.GetTotalVolumeNodeSamples();

      // apply changes
      // first proposal only
      if (proposal.first>=0 && proposal.second<0) {
        dim3 dimBlock(1,1,1);
        dim3 dimGrid(1,1,1);
        cudaUpdateVolume1<<<dimGrid, dimBlock>>>(cudaVolume,
                                                 proposal.first,
                                                 c%NUM_MATERIALS);
      }

      // apply changes
      // both proposals
      else if (proposal.first>=0 && proposal.second>=0) {
        dim3 dimBlock(1,1,1);
        dim3 dimGrid(1,1,1);
        cudaUpdateVolume2<<<dimGrid, dimBlock>>>(cudaVolume,
                                                 proposal.first,
                                                 c%NUM_MATERIALS,
                                                 proposal.second,
                                                 c/NUM_MATERIALS);
      }
    }


    // copy linear collection volumes to arrays
    for (int c=collectionStart; c<collectionEnd; c++) {
      float4 *cudaVolume = cudaVolumeLinearCollection[g] + c*mGeometry.GetTotalVolumeNodeSamples();

      hipArray **hipArray = NULL;
      switch (c) {
      case 0:
        hipArray = &cudaVolumeArray00[g];
        break;
      case 1:
        hipArray = &cudaVolumeArray01[g];
        break;
      case 2:
        hipArray = &cudaVolumeArray02[g];
        break;
      case 3:
        hipArray = &cudaVolumeArray10[g];
        break;
      case 4:
        hipArray = &cudaVolumeArray11[g];
        break;
      case 5:
        hipArray = &cudaVolumeArray12[g];
        break;
      case 6:
        hipArray = &cudaVolumeArray20[g];
        break;
      case 7:
        hipArray = &cudaVolumeArray21[g];
        break;
      case 8:
        hipArray = &cudaVolumeArray22[g];
        break;
      }

      // copy data to 3D array
      hipMemcpy3DParms copyParams = {0};
      copyParams.srcPtr   = make_hipPitchedPtr(cudaVolume, volumeSize.width*sizeof(float4), volumeSize.width, volumeSize.height);
      copyParams.dstArray = *hipArray;
      copyParams.extent   = volumeSize;
      copyParams.kind     = hipMemcpyDeviceToDevice;
      checkCudaErrors(hipMemcpy3DAsync(&copyParams));
    }
  }

  TIMER_STOP;
}


//==================================================================================================
//==================================================================================================
//==================================================================================================
void MarkovContext::CudaSetBaselineProjection() const {
  for (int g=0; g<mGpuIds.size(); g++) {
    checkCudaErrors(hipSetDevice(mGpuIds[g]));

    checkCudaErrors(hipMalloc(&cudaBaselineProjection[g], sizeof(float)*mBaselineProjection.size()));
    checkCudaErrors(hipMemcpy(cudaBaselineProjection[g], &mBaselineProjection[0], sizeof(float)*mBaselineProjection.size(), hipMemcpyHostToDevice));
  }
}


template <typename T>
void SetTextureParams(T *cudaTexture) {
  cudaTexture->normalized = true;
  cudaTexture->filterMode = hipFilterModeLinear;
  cudaTexture->addressMode[0] = hipAddressModeClamp;
  cudaTexture->addressMode[1] = hipAddressModeClamp;
  cudaTexture->addressMode[2] = hipAddressModeClamp;
}

void MarkovContext::CudaInitialize() {

  mGpuIds.clear();

  int gpu_bitfield = mGeometry.GetGPUBitfield();
  for (int g=0; g<MAX_GPUS; g++) {
    if ((1<<g)&gpu_bitfield) {
      int id = gpuDeviceInit(g);
      if (id>=0) {
        mGpuIds.push_back(id);

        // get device name
        hipDeviceProp_t deviceProps;
        checkCudaErrors(hipGetDeviceProperties(&deviceProps, id));
        printf("CUDA device [%s]\n", deviceProps.name);
      }
    }
  }

  if (mGpuIds.empty()) {
    std::cerr<<"No GPUs selected!!"<<std::endl;
  }


  for (int g=0; g<mGpuIds.size(); g++) {
    checkCudaErrors(hipSetDevice(mGpuIds[g]));

    // initialize material data
    for (int m=0; m<mMaterials.size(); m++) {
      vector<float> scatterFactors;
      mMaterials[m].GetScatterFactorArray(scatterFactors);

      // Allocate array and copy image data
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
      hipArray *cuArray;
      checkCudaErrors(hipMallocArray(&cuArray,
                                      &channelDesc,
                                      (int)scatterFactors.size(),
                                      1,
                                      hipArrayDefault));
      cudaMaterialArray[g][m] = cuArray;

      checkCudaErrors(hipMemcpyToArray(cuArray,
                                        0,
                                        0,
                                        &scatterFactors[0],
                                        (int)scatterFactors.size() * sizeof(float),
                                        hipMemcpyHostToDevice));


      texture<float, 1, hipReadModeElementType> *cudaMaterialTexture = NULL;
      switch (m) {
      case 0:
        cudaMaterialTexture = &cudaMaterialTextures0; break;
      case 1:
        cudaMaterialTexture = &cudaMaterialTextures1; break;
      case 2:
        cudaMaterialTexture = &cudaMaterialTextures2; break;
      }

      // Set texture parameters
      cudaMaterialTexture->addressMode[0] = hipAddressModeClamp;
      cudaMaterialTexture->addressMode[1] = hipAddressModeClamp;
      cudaMaterialTexture->filterMode = hipFilterModeLinear;
      cudaMaterialTexture->normalized = true;    // access with normalized texture coordinates
    
      // Bind the array to the texture
      checkCudaErrors(hipBindTextureToArray(*cudaMaterialTexture, cuArray, channelDesc));
    }


    // volume data
    hipExtent volumeSize = make_hipExtent(mGeometry.GetVolumeNodeSamplesX(),
                                            mGeometry.GetVolumeNodeSamplesY(),
                                            mGeometry.GetVolumeNodeSamplesZ());
    checkCudaErrors(hipMalloc(&cudaVolumeLinearCollection[g], sizeof(float4)*mGeometry.GetTotalVolumeNodeSamples()*NUM_MATERIALS*NUM_MATERIALS));
    hipChannelFormatDesc channelDesc4 = hipCreateChannelDesc<float4>();
    checkCudaErrors(hipMalloc3DArray(&cudaVolumeArray00[g], &channelDesc4, volumeSize));
    checkCudaErrors(hipMalloc3DArray(&cudaVolumeArray01[g], &channelDesc4, volumeSize));
    checkCudaErrors(hipMalloc3DArray(&cudaVolumeArray02[g], &channelDesc4, volumeSize));
    checkCudaErrors(hipMalloc3DArray(&cudaVolumeArray10[g], &channelDesc4, volumeSize));
    checkCudaErrors(hipMalloc3DArray(&cudaVolumeArray11[g], &channelDesc4, volumeSize));
    checkCudaErrors(hipMalloc3DArray(&cudaVolumeArray12[g], &channelDesc4, volumeSize));
    checkCudaErrors(hipMalloc3DArray(&cudaVolumeArray20[g], &channelDesc4, volumeSize));
    checkCudaErrors(hipMalloc3DArray(&cudaVolumeArray21[g], &channelDesc4, volumeSize));
    checkCudaErrors(hipMalloc3DArray(&cudaVolumeArray22[g], &channelDesc4, volumeSize));

    SetTextureParams(&cudaVolumeTextures00);
    SetTextureParams(&cudaVolumeTextures01);
    SetTextureParams(&cudaVolumeTextures02);
    SetTextureParams(&cudaVolumeTextures10);
    SetTextureParams(&cudaVolumeTextures11);
    SetTextureParams(&cudaVolumeTextures12);
    SetTextureParams(&cudaVolumeTextures20);
    SetTextureParams(&cudaVolumeTextures21);
    SetTextureParams(&cudaVolumeTextures22);
    checkCudaErrors(hipBindTextureToArray(cudaVolumeTextures00, cudaVolumeArray00[g], channelDesc4));
    checkCudaErrors(hipBindTextureToArray(cudaVolumeTextures01, cudaVolumeArray01[g], channelDesc4));
    checkCudaErrors(hipBindTextureToArray(cudaVolumeTextures02, cudaVolumeArray02[g], channelDesc4));
    checkCudaErrors(hipBindTextureToArray(cudaVolumeTextures10, cudaVolumeArray10[g], channelDesc4));
    checkCudaErrors(hipBindTextureToArray(cudaVolumeTextures11, cudaVolumeArray11[g], channelDesc4));
    checkCudaErrors(hipBindTextureToArray(cudaVolumeTextures12, cudaVolumeArray12[g], channelDesc4));
    checkCudaErrors(hipBindTextureToArray(cudaVolumeTextures20, cudaVolumeArray20[g], channelDesc4));
    checkCudaErrors(hipBindTextureToArray(cudaVolumeTextures21, cudaVolumeArray21[g], channelDesc4));
    checkCudaErrors(hipBindTextureToArray(cudaVolumeTextures22, cudaVolumeArray22[g], channelDesc4));


    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    checkCudaErrors(hipMalloc3DArray(&cudaSourceArray00[g], &channelDesc, volumeSize));
    checkCudaErrors(hipMalloc3DArray(&cudaSourceArray01[g], &channelDesc, volumeSize));
    checkCudaErrors(hipMalloc3DArray(&cudaSourceArray02[g], &channelDesc, volumeSize));
    checkCudaErrors(hipMalloc3DArray(&cudaSourceArray10[g], &channelDesc, volumeSize));
    checkCudaErrors(hipMalloc3DArray(&cudaSourceArray11[g], &channelDesc, volumeSize));
    checkCudaErrors(hipMalloc3DArray(&cudaSourceArray12[g], &channelDesc, volumeSize));
    checkCudaErrors(hipMalloc3DArray(&cudaSourceArray20[g], &channelDesc, volumeSize));
    checkCudaErrors(hipMalloc3DArray(&cudaSourceArray21[g], &channelDesc, volumeSize));
    checkCudaErrors(hipMalloc3DArray(&cudaSourceArray22[g], &channelDesc, volumeSize));

    SetTextureParams(&cudaSourceTextures00);
    SetTextureParams(&cudaSourceTextures01);
    SetTextureParams(&cudaSourceTextures02);
    SetTextureParams(&cudaSourceTextures10);
    SetTextureParams(&cudaSourceTextures11);
    SetTextureParams(&cudaSourceTextures12);
    SetTextureParams(&cudaSourceTextures20);
    SetTextureParams(&cudaSourceTextures21);
    SetTextureParams(&cudaSourceTextures22);
    checkCudaErrors(hipBindTextureToArray(cudaSourceTextures00, cudaSourceArray00[g], channelDesc));
    checkCudaErrors(hipBindTextureToArray(cudaSourceTextures01, cudaSourceArray01[g], channelDesc));
    checkCudaErrors(hipBindTextureToArray(cudaSourceTextures02, cudaSourceArray02[g], channelDesc));
    checkCudaErrors(hipBindTextureToArray(cudaSourceTextures10, cudaSourceArray10[g], channelDesc));
    checkCudaErrors(hipBindTextureToArray(cudaSourceTextures11, cudaSourceArray11[g], channelDesc));
    checkCudaErrors(hipBindTextureToArray(cudaSourceTextures12, cudaSourceArray12[g], channelDesc));
    checkCudaErrors(hipBindTextureToArray(cudaSourceTextures20, cudaSourceArray20[g], channelDesc));
    checkCudaErrors(hipBindTextureToArray(cudaSourceTextures21, cudaSourceArray21[g], channelDesc));
    checkCudaErrors(hipBindTextureToArray(cudaSourceTextures22, cudaSourceArray22[g], channelDesc));


    // upload all of the ray info
    checkCudaErrors(hipMalloc(&cudaSourceRayVolOrigin[g], sizeof(float3)*mSourceRayVolOrigin.size()));
    checkCudaErrors(hipMemcpy(cudaSourceRayVolOrigin[g], &mSourceRayVolOrigin[0], sizeof(float3)*mSourceRayVolOrigin.size(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc(&cudaSourceRayVolDir[g], sizeof(float3)*mSourceRayVolDir.size()));
    checkCudaErrors(hipMemcpy(cudaSourceRayVolDir[g], &mSourceRayVolDir[0], sizeof(float3)*mSourceRayVolDir.size(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc(&cudaSourceRayTMin[g], sizeof(float)*mSourceRayTMin.size()));
    checkCudaErrors(hipMemcpy(cudaSourceRayTMin[g], &mSourceRayTMin[0], sizeof(float)*mSourceRayTMin.size(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc(&cudaSourceRayTMax[g], sizeof(float)*mSourceRayTMax.size()));
    checkCudaErrors(hipMemcpy(cudaSourceRayTMax[g], &mSourceRayTMax[0], sizeof(float)*mSourceRayTMax.size(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc(&cudaDetectorRayVolOrigin[g], sizeof(float3)*mDetectorRayVolOrigin.size()));
    checkCudaErrors(hipMemcpy(cudaDetectorRayVolOrigin[g], &mDetectorRayVolOrigin[0], sizeof(float3)*mDetectorRayVolOrigin.size(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc(&cudaDetectorRayVolDir[g], sizeof(float3)*mDetectorRayVolDir.size()));
    checkCudaErrors(hipMemcpy(cudaDetectorRayVolDir[g], &mDetectorRayVolDir[0], sizeof(float3)*mDetectorRayVolDir.size(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc(&cudaDetectorRayWorldOrigin[g], sizeof(float3)*mDetectorRayOrigin.size()));
    checkCudaErrors(hipMemcpy(cudaDetectorRayWorldOrigin[g], &mDetectorRayOrigin[0], sizeof(float3)*mDetectorRayOrigin.size(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc(&cudaDetectorRayWorldDir[g], sizeof(float3)*mDetectorRayDir.size()));
    checkCudaErrors(hipMemcpy(cudaDetectorRayWorldDir[g], &mDetectorRayDir[0], sizeof(float3)*mDetectorRayDir.size(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc(&cudaDetectorRayTMin[g], sizeof(float)*mDetectorRayTMin.size()));
    checkCudaErrors(hipMemcpy(cudaDetectorRayTMin[g], &mDetectorRayTMin[0], sizeof(float)*mDetectorRayTMin.size(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc(&cudaDetectorRayTMax[g], sizeof(float)*mDetectorRayTMax.size()));
    checkCudaErrors(hipMemcpy(cudaDetectorRayTMax[g], &mDetectorRayTMax[0], sizeof(float)*mDetectorRayTMax.size(), hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc(&cudaRayIds[g], sizeof(unsigned int)*mGeometry.GetTotalProjectionSamples()));
    checkCudaErrors(hipMalloc(&cudaRayPriority[g], sizeof(float)*mGeometry.GetTotalProjectionSamples()));
    checkCudaErrors(hipMalloc(&cudaCurrentForwardProjection[g], sizeof(float)*mGeometry.GetTotalProjectionSamples()));

    vector<float> projectionAngles;
    for (int i=0; i<mGeometry.GetNumProjectionAngles(); i++) {
      projectionAngles.push_back(mGeometry.GetProjectionAngle(i));
    }
    checkCudaErrors(hipMalloc(&cudaProjectionAngles[g], sizeof(float)*mGeometry.GetNumProjectionAngles()));
    checkCudaErrors(hipMemcpy(cudaProjectionAngles[g], &projectionAngles[0], sizeof(float)*mGeometry.GetNumProjectionAngles(), hipMemcpyHostToDevice));


    // precompute some source attenuation info
    vector<float> sourceScale(mGeometry.GetTotalVolumeNodeSamples());
    for (int nvi=0; nvi<mGeometry.GetTotalVolumeNodeSamples(); nvi++) {
      int x,y,z;
      mGeometry.VolumeIndexToNodeCoord(nvi, x,y,z);

      Vec3f voxelPosition;
      mGeometry.VolumeToWorld(Vec3f((float)x,(float)y,(float)z), voxelPosition);

      Vec3f diff = voxelPosition - mGeometry.GetSourcePosition();
      float maxt = diff.Length();
      Vec3f dir = diff / maxt;
    
      sourceScale[nvi] = 1 / (maxt*maxt);
    }

    checkCudaErrors(hipMalloc(&cudaSourceScale[g], sizeof(float)*sourceScale.size()));
    checkCudaErrors(hipMemcpy(cudaSourceScale[g], &sourceScale[0], sizeof(float)*sourceScale.size(), hipMemcpyHostToDevice));


    // source falloff texture
    hipChannelFormatDesc falloffChannelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    checkCudaErrors(hipMallocArray(&cudaFalloffArray[g],
                                    &falloffChannelDesc,
                                    mGeometry.GetSourceAttenMapWidth(),
                                    mGeometry.GetSourceAttenMapHeight(),
                                    hipArrayDefault));
    checkCudaErrors(hipMemcpyToArray(cudaFalloffArray[g],
                                      0,
                                      0,
                                      &mGeometry.GetSourceAttenMap()[0],
                                      mGeometry.GetSourceAttenMapWidth()*mGeometry.GetSourceAttenMapHeight() * sizeof(float),
                                      hipMemcpyHostToDevice));
    cudaSourceFalloffTexture.addressMode[0] = hipAddressModeClamp;
    cudaSourceFalloffTexture.addressMode[1] = hipAddressModeClamp;
    cudaSourceFalloffTexture.filterMode = hipFilterModeLinear;
    cudaSourceFalloffTexture.normalized = true;    // access with normalized texture coordinates
    checkCudaErrors(hipBindTextureToArray(cudaSourceFalloffTexture, cudaFalloffArray[g], falloffChannelDesc));



    checkCudaErrors(hipMalloc(&cudaSourceAttenuationCollection[g], sizeof(float)*mGeometry.GetTotalVolumeNodeSamples() * NUM_MATERIALS*NUM_MATERIALS));
    checkCudaErrors(hipMalloc(&cudaForwardProjectionCollection[g], sizeof(float)*mGeometry.GetTotalProjectionSamples() * NUM_MATERIALS*NUM_MATERIALS));
    checkCudaErrors(hipMalloc(&cudaForwardProjectionError[g], sizeof(float)*mGeometry.GetTotalProjectionSamples() * NUM_MATERIALS*NUM_MATERIALS));
  }
}


void MarkovContext::CudaShutdown() const {
  for (int g=0; g<mGpuIds.size(); g++) {
    checkCudaErrors(hipSetDevice(mGpuIds[g]));
    
    checkCudaErrors(hipFree(cudaBaselineProjection[g]));  cudaBaselineProjection[g]=NULL;

    for (int m=0; m<mMaterials.size(); m++) {
      checkCudaErrors(hipFree(cudaMaterialArray[g][m]));  cudaMaterialArray[g][m]=NULL;
    }

    checkCudaErrors(hipFree(cudaVolumeLinearCollection[g]));  cudaVolumeLinearCollection[g]=NULL;

    checkCudaErrors(hipFreeArray(cudaVolumeArray00[g]));  cudaVolumeArray00[g]=NULL;
    checkCudaErrors(hipFreeArray(cudaVolumeArray01[g]));  cudaVolumeArray01[g]=NULL;
    checkCudaErrors(hipFreeArray(cudaVolumeArray02[g]));  cudaVolumeArray02[g]=NULL;
    checkCudaErrors(hipFreeArray(cudaVolumeArray10[g]));  cudaVolumeArray10[g]=NULL;
    checkCudaErrors(hipFreeArray(cudaVolumeArray11[g]));  cudaVolumeArray11[g]=NULL;
    checkCudaErrors(hipFreeArray(cudaVolumeArray12[g]));  cudaVolumeArray12[g]=NULL;
    checkCudaErrors(hipFreeArray(cudaVolumeArray20[g]));  cudaVolumeArray20[g]=NULL;
    checkCudaErrors(hipFreeArray(cudaVolumeArray21[g]));  cudaVolumeArray21[g]=NULL;
    checkCudaErrors(hipFreeArray(cudaVolumeArray22[g]));  cudaVolumeArray22[g]=NULL;
    checkCudaErrors(hipFreeArray(cudaSourceArray00[g]));  cudaSourceArray00[g]=NULL;
    checkCudaErrors(hipFreeArray(cudaSourceArray01[g]));  cudaSourceArray01[g]=NULL;
    checkCudaErrors(hipFreeArray(cudaSourceArray02[g]));  cudaSourceArray02[g]=NULL;
    checkCudaErrors(hipFreeArray(cudaSourceArray10[g]));  cudaSourceArray10[g]=NULL;
    checkCudaErrors(hipFreeArray(cudaSourceArray11[g]));  cudaSourceArray11[g]=NULL;
    checkCudaErrors(hipFreeArray(cudaSourceArray12[g]));  cudaSourceArray12[g]=NULL;
    checkCudaErrors(hipFreeArray(cudaSourceArray20[g]));  cudaSourceArray20[g]=NULL;
    checkCudaErrors(hipFreeArray(cudaSourceArray21[g]));  cudaSourceArray21[g]=NULL;
    checkCudaErrors(hipFreeArray(cudaSourceArray22[g]));  cudaSourceArray22[g]=NULL;

    checkCudaErrors(hipFree(cudaSourceRayVolOrigin[g]));  cudaSourceRayVolOrigin[g]=NULL;
    checkCudaErrors(hipFree(cudaSourceRayVolDir[g]));  cudaSourceRayVolDir[g]=NULL;
    checkCudaErrors(hipFree(cudaSourceRayTMin[g]));  cudaSourceRayTMin[g]=NULL;
    checkCudaErrors(hipFree(cudaSourceRayTMax[g]));  cudaSourceRayTMax[g]=NULL;
    checkCudaErrors(hipFree(cudaDetectorRayVolOrigin[g]));  cudaDetectorRayVolOrigin[g]=NULL;
    checkCudaErrors(hipFree(cudaDetectorRayVolDir[g]));  cudaDetectorRayVolDir[g]=NULL;
    checkCudaErrors(hipFree(cudaDetectorRayWorldOrigin[g]));  cudaDetectorRayWorldOrigin[g]=NULL;
    checkCudaErrors(hipFree(cudaDetectorRayWorldDir[g]));  cudaDetectorRayWorldDir[g]=NULL;
    checkCudaErrors(hipFree(cudaDetectorRayTMin[g]));  cudaDetectorRayTMin[g]=NULL;
    checkCudaErrors(hipFree(cudaDetectorRayTMax[g]));  cudaDetectorRayTMax[g]=NULL;

    checkCudaErrors(hipFree(cudaRayIds[g]));  cudaRayIds[g]=NULL;
    checkCudaErrors(hipFree(cudaRayPriority[g]));  cudaRayPriority[g]=NULL;
    checkCudaErrors(hipFree(cudaCurrentForwardProjection[g]));  cudaCurrentForwardProjection[g]=NULL;

    checkCudaErrors(hipFree(cudaProjectionAngles[g]));  cudaProjectionAngles[g]=NULL;

    checkCudaErrors(hipFree(cudaSourceScale[g]));  cudaSourceScale[g]=NULL;

    checkCudaErrors(hipFreeArray(cudaFalloffArray[g]));  cudaFalloffArray[g]=NULL;

    checkCudaErrors(hipFree(cudaSourceAttenuationCollection[g]));  cudaSourceAttenuationCollection[g]=NULL;
    checkCudaErrors(hipFree(cudaForwardProjectionCollection[g]));  cudaForwardProjectionCollection[g]=NULL;
    checkCudaErrors(hipFree(cudaForwardProjectionError[g]));  cudaForwardProjectionError[g]=NULL;

    hipDeviceReset();
  }
}


void MarkovContext::CudaGetCollectionStartEnd(int g, int collectionSize,
                                              int &start, int &end) const {
  // default
  start = g * collectionSize / mGpuIds.size();
  end = (g+1) * collectionSize / mGpuIds.size();

  if (mGpuIds.size() == 1) {
    start = 0;
    end = collectionSize;
  }

  else if (mGpuIds.size() == 2) {
    if (collectionSize == 1) {
      if (g==0) {
        start = 0;
        end = 0;
      }
      else {
        start = 0;
        end = 1;
      }
    }

    else if (collectionSize == 9) {
      if (g==0) {
        start = 0;
        end = 4;
      }
      else {
        start = 4;
        end = 9;
      }
    }
  }
}
